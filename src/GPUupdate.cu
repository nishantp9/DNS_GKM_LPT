#include "hip/hip_runtime.h"
/**************************************************************************************************************************
 * ----------------------------------     Updating Flow-Field on GPU   ---------------------------------------------------*
 * -----------------------------------------------------------------------------------------------------------------------*
 **************************************************************************************************************************/
#include "GPUupdate.h"


/** -----------------------------------------*
 * HANDLING ERROR 
 * ------------------------------------------*/
/*
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
    file, line );
    exit( EXIT_FAILURE );
  }
}
*/

/* 
 * global : GPU GLOBAL FUNCTION 
 * device : GPU DEVICE FUNCTION (Called from global function)
 */
__global__ void derivsX_Kernel(ptype *d_W0, ptype *d_Wl0, ptype *d_Wr0, ptype *d_DW0xl, ptype *d_DW0yl, ptype *d_DW0zl, ptype *d_DW0xr, ptype *d_DW0yr, ptype *d_DW0zr,
							   ptype *d_W1, ptype *d_Wl1, ptype *d_Wr1, ptype *d_DW1xl, ptype *d_DW1yl, ptype *d_DW1zl, ptype *d_DW1xr, ptype *d_DW1yr, ptype *d_DW1zr,
							   ptype *d_W2, ptype *d_Wl2, ptype *d_Wr2, ptype *d_DW2xl, ptype *d_DW2yl, ptype *d_DW2zl, ptype *d_DW2xr, ptype *d_DW2yr, ptype *d_DW2zr,
							   ptype *d_W3, ptype *d_Wl3, ptype *d_Wr3, ptype *d_DW3xl, ptype *d_DW3yl, ptype *d_DW3zl, ptype *d_DW3xr, ptype *d_DW3yr, ptype *d_DW3zr,
							   ptype *d_W4, ptype *d_Wl4, ptype *d_Wr4, ptype *d_DW4xl, ptype *d_DW4yl, ptype *d_DW4zl, ptype *d_DW4xr, ptype *d_DW4yr, ptype *d_DW4zr);

__global__ void derivsY_Kernel(ptype *d_W0, ptype *d_Wl0, ptype *d_Wr0, ptype *d_DW0xl, ptype *d_DW0yl, ptype *d_DW0zl, ptype *d_DW0xr, ptype *d_DW0yr, ptype *d_DW0zr,
							   ptype *d_W1, ptype *d_Wl1, ptype *d_Wr1, ptype *d_DW1xl, ptype *d_DW1yl, ptype *d_DW1zl, ptype *d_DW1xr, ptype *d_DW1yr, ptype *d_DW1zr,
							   ptype *d_W2, ptype *d_Wl2, ptype *d_Wr2, ptype *d_DW2xl, ptype *d_DW2yl, ptype *d_DW2zl, ptype *d_DW2xr, ptype *d_DW2yr, ptype *d_DW2zr,
							   ptype *d_W3, ptype *d_Wl3, ptype *d_Wr3, ptype *d_DW3xl, ptype *d_DW3yl, ptype *d_DW3zl, ptype *d_DW3xr, ptype *d_DW3yr, ptype *d_DW3zr,
							   ptype *d_W4, ptype *d_Wl4, ptype *d_Wr4, ptype *d_DW4xl, ptype *d_DW4yl, ptype *d_DW4zl, ptype *d_DW4xr, ptype *d_DW4yr, ptype *d_DW4zr);

__global__ void derivsZ_Kernel(ptype *d_W0, ptype *d_Wl0, ptype *d_Wr0, ptype *d_DW0xl, ptype *d_DW0yl, ptype *d_DW0zl, ptype *d_DW0xr, ptype *d_DW0yr, ptype *d_DW0zr,
							   ptype *d_W1, ptype *d_Wl1, ptype *d_Wr1, ptype *d_DW1xl, ptype *d_DW1yl, ptype *d_DW1zl, ptype *d_DW1xr, ptype *d_DW1yr, ptype *d_DW1zr,
							   ptype *d_W2, ptype *d_Wl2, ptype *d_Wr2, ptype *d_DW2xl, ptype *d_DW2yl, ptype *d_DW2zl, ptype *d_DW2xr, ptype *d_DW2yr, ptype *d_DW2zr,
							   ptype *d_W3, ptype *d_Wl3, ptype *d_Wr3, ptype *d_DW3xl, ptype *d_DW3yl, ptype *d_DW3zl, ptype *d_DW3xr, ptype *d_DW3yr, ptype *d_DW3zr,
							   ptype *d_W4, ptype *d_Wl4, ptype *d_Wr4, ptype *d_DW4xl, ptype *d_DW4yl, ptype *d_DW4zl, ptype *d_DW4xr, ptype *d_DW4yr, ptype *d_DW4zr);

__global__ void flip_Kernel(ptype *d_Wl1, ptype *d_Wl2, ptype *d_DW1xl, ptype *d_DW2xl, ptype *d_DW1yl, ptype *d_DW2yl, ptype *d_DW1zl, ptype *d_DW2zl,
                            ptype *d_Wr1, ptype *d_Wr2, ptype *d_DW1xr, ptype *d_DW2xr, ptype *d_DW1yr, ptype *d_DW2yr, ptype *d_DW1zr, ptype *d_DW2zr);

__global__ void flipBack_Kernel(ptype *d_F1, ptype *d_F2);

__global__ void update_Kernel (ptype *d_W, ptype *d_Fx, ptype *d_Fy, ptype *d_Fz);								  
										  
__global__ void flux(ptype *d_W0, ptype *d_W1, ptype *d_W2, ptype *d_W3, ptype *d_W4,
					 ptype *d_Wl0, ptype *d_Wl1, ptype *d_Wl2, ptype *d_Wl3, ptype *d_Wl4,
					 ptype *d_Wr0, ptype *d_Wr1, ptype *d_Wr2, ptype *d_Wr3, ptype *d_Wr4,
					 ptype *d_DW0xl, ptype *d_DW1xl, ptype *d_DW2xl, ptype *d_DW3xl, ptype *d_DW4xl,
					 ptype *d_DW0xr, ptype *d_DW1xr, ptype *d_DW2xr, ptype *d_DW3xr, ptype *d_DW4xr,
					 ptype *d_DW0yl, ptype *d_DW1yl, ptype *d_DW2yl, ptype *d_DW3yl, ptype *d_DW4yl,
					 ptype *d_DW0yr, ptype *d_DW1yr, ptype *d_DW2yr, ptype *d_DW3yr, ptype *d_DW4yr,
					 ptype *d_DW0zl, ptype *d_DW1zl, ptype *d_DW2zl, ptype *d_DW3zl, ptype *d_DW4zl,
					 ptype *d_DW0zr, ptype *d_DW1zr, ptype *d_DW2zr, ptype *d_DW3zr, ptype *d_DW4zr,
					 ptype *d_F0, ptype *d_F1, ptype *d_F2, ptype *d_F3, ptype *d_F4, int TAG);

__device__ void d_slopesolver(ptype b[5], ptype U[3], ptype lam, ptype a[5]);
__device__ void d_MCal(ptype M[5], ptype I[3][7], ptype If[3][7], ptype Ie2, ptype Ie4, int k, int l, int m, ptype ax[5]);
__device__ void d_MCal(ptype M[5], ptype I[3][7], ptype If[3][7], ptype Ie2, ptype Ie4, int k, int l, int m);

__device__ void d_c2p(ptype W[5], ptype &den, ptype U[3], ptype &P);

__device__ void d_ApplyWENO(ptype WLLL[5], ptype WLL[5], ptype WL[5], ptype WR[5], ptype WRR[5],  ptype WRRR[5], ptype Wl[5], ptype Wr[5]);

__global__ void W2T3D(ptype *d_W0, ptype *d_W1, ptype *d_W2, ptype *d_W3, ptype *d_W4, ptype *d_T);

__constant__ int d_nt_segx, d_nc_segx, d_nt_segy, d_nc_segy, d_nt_segz, d_nc_segz, d_Nt_seg, d_Nc_seg, d_K;

__constant__ ptype d_dx, d_dt, d_mu0, d_T0, GAM;

/**----------------------------------------------------------------------------------------------------------------------*
 * Function : Evolves Flow Field over the whole 3-D domain
 *-----------------------------------------------------------------------------------------------------------------------*/
void evolve(ptype *W[5], DevAlloc *dev, ptype dt)
{
	//hipSetDevice(int(myrank_3d%2));
	int blkz  = (Nt_seg-1)/Bsz3D + 1;
	int blkf  = (Nt_seg-1)/Bszf  + 1;
	int blkst = (Nt_seg-1)/Bszs  + 1;

/* ----------------------------------------------------*
 * 	Host (CPU) to Device (GPU) Flow Field data Transfer
 * ----------------------------------------------------*/
	HtoD(dev->d_W0, W[0]);
	HtoD(dev->d_W1, W[1]); 
	HtoD(dev->d_W2, W[2]); 
	HtoD(dev->d_W3, W[3]); 
	HtoD(dev->d_W4, W[4]); 
		
	hipDeviceSynchronize();
	
	ptype Dx = dx;
	ptype Mu = mu0;
	ptype T	= T0;	
	ptype Ga = gam;
	ptype Dt = dt;
	(hipMemcpyToSymbol(HIP_SYMBOL(d_nt_segx), &nt_segx, sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_nc_segx), &nc_segx, sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_nt_segy), &nt_segy, sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_nc_segy), &nc_segy, sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_nt_segz), &nt_segz, sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_nc_segz), &nc_segz, sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_Nt_seg),  &Nt_seg, sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_Nc_seg),  &Nc_seg, sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_K),       &K,      sizeof(int)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_dx),      &Dx, sizeof(ptype)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_dt),      &Dt, sizeof(ptype)));	
	(hipMemcpyToSymbol(HIP_SYMBOL(d_mu0),     &Mu, sizeof(ptype)));
	(hipMemcpyToSymbol(HIP_SYMBOL(GAM),       &Ga, sizeof(ptype)));
	(hipMemcpyToSymbol(HIP_SYMBOL(d_T0),      &T,  sizeof(ptype)));
	
	hipDeviceSynchronize();
	
	derivsX_Kernel<<<blkst, Bszs>>>(dev->d_W0, dev->d_Wl0, dev->d_Wr0, dev->d_DW0xl, dev->d_DW0yl, dev->d_DW0zl, dev->d_DW0xr, dev->d_DW0yr, dev->d_DW0zr,
									dev->d_W1, dev->d_Wl1, dev->d_Wr1, dev->d_DW1xl, dev->d_DW1yl, dev->d_DW1zl, dev->d_DW1xr, dev->d_DW1yr, dev->d_DW1zr,
									dev->d_W2, dev->d_Wl2, dev->d_Wr2, dev->d_DW2xl, dev->d_DW2yl, dev->d_DW2zl, dev->d_DW2xr, dev->d_DW2yr, dev->d_DW2zr,
									dev->d_W3, dev->d_Wl3, dev->d_Wr3, dev->d_DW3xl, dev->d_DW3yl, dev->d_DW3zl, dev->d_DW3xr, dev->d_DW3yr, dev->d_DW3zr,
									dev->d_W4, dev->d_Wl4, dev->d_Wr4, dev->d_DW4xl, dev->d_DW4yl, dev->d_DW4zl, dev->d_DW4xr, dev->d_DW4yr, dev->d_DW4zr);

	hipDeviceSynchronize();

	flux<<<blkf, Bszf>>>(dev->d_W0, dev->d_W1, dev->d_W2, dev->d_W3, dev->d_W4,
						 dev->d_Wl0, dev->d_Wl1, dev->d_Wl2, dev->d_Wl3, dev->d_Wl4,
						 dev->d_Wr0, dev->d_Wr1, dev->d_Wr2, dev->d_Wr3, dev->d_Wr4,
						 dev->d_DW0xl, dev->d_DW1xl, dev->d_DW2xl, dev->d_DW3xl, dev->d_DW4xl,
						 dev->d_DW0xr, dev->d_DW1xr, dev->d_DW2xr, dev->d_DW3xr, dev->d_DW4xr,  
						 dev->d_DW0yl, dev->d_DW1yl, dev->d_DW2yl, dev->d_DW3yl, dev->d_DW4yl, 
						 dev->d_DW0yr, dev->d_DW1yr, dev->d_DW2yr, dev->d_DW3yr, dev->d_DW4yr, 
						 dev->d_DW0zl, dev->d_DW1zl, dev->d_DW2zl, dev->d_DW3zl, dev->d_DW4zl, 
						 dev->d_DW0zr, dev->d_DW1zr, dev->d_DW2zr, dev->d_DW3zr, dev->d_DW4zr, 
						 dev->d_F0x, dev->d_F1x, dev->d_F2x, dev->d_F3x, dev->d_F4x, 1);
	
	hipDeviceSynchronize();
	
	derivsY_Kernel<<<blkst, Bszs>>>(dev->d_W0, dev->d_Wl0, dev->d_Wr0, dev->d_DW0xl, dev->d_DW0yl, dev->d_DW0zl, dev->d_DW0xr, dev->d_DW0yr, dev->d_DW0zr,
									dev->d_W1, dev->d_Wl1, dev->d_Wr1, dev->d_DW1xl, dev->d_DW1yl, dev->d_DW1zl, dev->d_DW1xr, dev->d_DW1yr, dev->d_DW1zr,
									dev->d_W2, dev->d_Wl2, dev->d_Wr2, dev->d_DW2xl, dev->d_DW2yl, dev->d_DW2zl, dev->d_DW2xr, dev->d_DW2yr, dev->d_DW2zr,
									dev->d_W3, dev->d_Wl3, dev->d_Wr3, dev->d_DW3xl, dev->d_DW3yl, dev->d_DW3zl, dev->d_DW3xr, dev->d_DW3yr, dev->d_DW3zr,
									dev->d_W4, dev->d_Wl4, dev->d_Wr4, dev->d_DW4xl, dev->d_DW4yl, dev->d_DW4zl, dev->d_DW4xr, dev->d_DW4yr, dev->d_DW4zr);

	hipDeviceSynchronize();
		
	
	flip_Kernel<<<blkz, Bsz3D>>>(dev->d_Wl1, dev->d_Wl2, dev->d_DW1xl, dev->d_DW2xl, dev->d_DW1yl, dev->d_DW2yl, dev->d_DW1zl, dev->d_DW2zl, 
	                             dev->d_Wr1, dev->d_Wr2, dev->d_DW1xr, dev->d_DW2xr, dev->d_DW1yr, dev->d_DW2yr, dev->d_DW1zr, dev->d_DW2zr);
	(hipDeviceSynchronize());
					
	flux<<<blkf, Bszf>>>(dev->d_W0, dev->d_W1, dev->d_W2, dev->d_W3, dev->d_W4,
						 dev->d_Wl0, dev->d_Wl1, dev->d_Wl2, dev->d_Wl3, dev->d_Wl4,
						 dev->d_Wr0, dev->d_Wr1, dev->d_Wr2, dev->d_Wr3, dev->d_Wr4,
						 dev->d_DW0xl, dev->d_DW1xl, dev->d_DW2xl, dev->d_DW3xl, dev->d_DW4xl,
						 dev->d_DW0xr, dev->d_DW1xr, dev->d_DW2xr, dev->d_DW3xr, dev->d_DW4xr,  
						 dev->d_DW0yl, dev->d_DW1yl, dev->d_DW2yl, dev->d_DW3yl, dev->d_DW4yl, 
						 dev->d_DW0yr, dev->d_DW1yr, dev->d_DW2yr, dev->d_DW3yr, dev->d_DW4yr, 
						 dev->d_DW0zl, dev->d_DW1zl, dev->d_DW2zl, dev->d_DW3zl, dev->d_DW4zl, 
						 dev->d_DW0zr, dev->d_DW1zr, dev->d_DW2zr, dev->d_DW3zr, dev->d_DW4zr, 
						 dev->d_F0y, dev->d_F1y, dev->d_F2y, dev->d_F3y, dev->d_F4y, 2);
			
	hipDeviceSynchronize();

	flipBack_Kernel<<<blkz, Bsz3D>>>(dev->d_F1y, dev->d_F2y);
					
	hipDeviceSynchronize();
			
	derivsZ_Kernel<<<blkst, Bszs>>>(dev->d_W0, dev->d_Wl0, dev->d_Wr0, dev->d_DW0xl, dev->d_DW0yl, dev->d_DW0zl, dev->d_DW0xr, dev->d_DW0yr, dev->d_DW0zr,
									dev->d_W1, dev->d_Wl1, dev->d_Wr1, dev->d_DW1xl, dev->d_DW1yl, dev->d_DW1zl, dev->d_DW1xr, dev->d_DW1yr, dev->d_DW1zr,
									dev->d_W2, dev->d_Wl2, dev->d_Wr2, dev->d_DW2xl, dev->d_DW2yl, dev->d_DW2zl, dev->d_DW2xr, dev->d_DW2yr, dev->d_DW2zr,
									dev->d_W3, dev->d_Wl3, dev->d_Wr3, dev->d_DW3xl, dev->d_DW3yl, dev->d_DW3zl, dev->d_DW3xr, dev->d_DW3yr, dev->d_DW3zr,
									dev->d_W4, dev->d_Wl4, dev->d_Wr4, dev->d_DW4xl, dev->d_DW4yl, dev->d_DW4zl, dev->d_DW4xr, dev->d_DW4yr, dev->d_DW4zr);

	hipDeviceSynchronize();
		
	flip_Kernel<<<blkz, Bsz3D>>>(dev->d_Wl1, dev->d_Wl3, dev->d_DW1xl, dev->d_DW3xl, dev->d_DW1yl, dev->d_DW3yl, dev->d_DW1zl, dev->d_DW3zl, 
								 dev->d_Wr1, dev->d_Wr3, dev->d_DW1xr, dev->d_DW3xr, dev->d_DW1yr, dev->d_DW3yr, dev->d_DW1zr, dev->d_DW3zr);

	hipDeviceSynchronize();
					
	flux<<<blkf, Bszf>>>(dev->d_W0, dev->d_W1, dev->d_W2, dev->d_W3, dev->d_W4,
						 dev->d_Wl0, dev->d_Wl1, dev->d_Wl2, dev->d_Wl3, dev->d_Wl4,
						 dev->d_Wr0, dev->d_Wr1, dev->d_Wr2, dev->d_Wr3, dev->d_Wr4,
						 dev->d_DW0xl, dev->d_DW1xl, dev->d_DW2xl, dev->d_DW3xl, dev->d_DW4xl,
						 dev->d_DW0xr, dev->d_DW1xr, dev->d_DW2xr, dev->d_DW3xr, dev->d_DW4xr,  
						 dev->d_DW0yl, dev->d_DW1yl, dev->d_DW2yl, dev->d_DW3yl, dev->d_DW4yl, 
						 dev->d_DW0yr, dev->d_DW1yr, dev->d_DW2yr, dev->d_DW3yr, dev->d_DW4yr, 
						 dev->d_DW0zl, dev->d_DW1zl, dev->d_DW2zl, dev->d_DW3zl, dev->d_DW4zl, 
						 dev->d_DW0zr, dev->d_DW1zr, dev->d_DW2zr, dev->d_DW3zr, dev->d_DW4zr, 
						 dev->d_F0z, dev->d_F1z, dev->d_F2z, dev->d_F3z, dev->d_F4z, 3);
	hipDeviceSynchronize();

	flipBack_Kernel<<<blkz, Bsz3D>>>(dev->d_F1z, dev->d_F3z);
																					
	hipDeviceSynchronize();

	update_Kernel<<<blkst, Bszs>>>(dev->d_W0, dev->d_F0x, dev->d_F0y, dev->d_F0z);
	update_Kernel<<<blkst, Bszs>>>(dev->d_W1, dev->d_F1x, dev->d_F1y, dev->d_F1z);
	update_Kernel<<<blkst, Bszs>>>(dev->d_W2, dev->d_F2x, dev->d_F2y, dev->d_F2z);
	update_Kernel<<<blkst, Bszs>>>(dev->d_W3, dev->d_F3x, dev->d_F3y, dev->d_F3z);
	update_Kernel<<<blkst, Bszs>>>(dev->d_W4, dev->d_F4x, dev->d_F4y, dev->d_F4z);
	
	hipDeviceSynchronize();
					
/* -------------------------------------------------------*
 * 	Applying Peridic BC and Printing Turbulent stats
 * -------------------------------------------------------*/
	DtoH(dev->d_W0, W[0]);	
	DtoH(dev->d_W1, W[1]); 	
	DtoH(dev->d_W2, W[2]);	
	DtoH(dev->d_W3, W[3]);
	DtoH(dev->d_W4, W[4]);

	hipDeviceSynchronize();
/** ---------------------------------------------------------------------------------------------------*
 * 	Main Iteration Loop ENDS  [Field Evolved]
 * ----------------------------------------------------------------------------------------------------*/	
}


/**-----------------------------------------------------------------------------------------*
 * Global Function : Calculates Derivatives required for calculation of flux in X-direction
 *------------------------------------------------------------------------------------------*/

__global__ void derivsX_Kernel(ptype *d_W0, ptype *d_Wl0, ptype *d_Wr0, ptype *d_DW0xl, ptype *d_DW0yl, ptype *d_DW0zl, ptype *d_DW0xr, ptype *d_DW0yr, ptype *d_DW0zr,
							   ptype *d_W1, ptype *d_Wl1, ptype *d_Wr1, ptype *d_DW1xl, ptype *d_DW1yl, ptype *d_DW1zl, ptype *d_DW1xr, ptype *d_DW1yr, ptype *d_DW1zr,
							   ptype *d_W2, ptype *d_Wl2, ptype *d_Wr2, ptype *d_DW2xl, ptype *d_DW2yl, ptype *d_DW2zl, ptype *d_DW2xr, ptype *d_DW2yr, ptype *d_DW2zr,
							   ptype *d_W3, ptype *d_Wl3, ptype *d_Wr3, ptype *d_DW3xl, ptype *d_DW3yl, ptype *d_DW3zl, ptype *d_DW3xr, ptype *d_DW3yr, ptype *d_DW3zr,
							   ptype *d_W4, ptype *d_Wl4, ptype *d_Wr4, ptype *d_DW4xl, ptype *d_DW4yl, ptype *d_DW4zl, ptype *d_DW4xr, ptype *d_DW4yr, ptype *d_DW4zr)
{
	int I = blockIdx.x*(blockDim.x) + threadIdx.x;

	int iy =  I/(d_nt_segx*d_nt_segz);
	int ix =  (I%(d_nt_segx*d_nt_segz))/d_nt_segz;
	int iz =  I%d_nt_segz;

	ptype WLLL[5], WLL[5], WL[5], WR[5], WRR[5], WRRR[5];
	ptype WLLLN[5], WLLN[5], WLN[5], WRN[5], WRRN[5], WRRRN[5];
	ptype WLLLS[5], WLLS[5], WLS[5], WRS[5], WRRS[5], WRRRS[5];
	ptype WLLLF[5], WLLF[5], WLF[5], WRF[5], WRRF[5], WRRRF[5];
	ptype WLLLB[5], WLLB[5], WLB[5], WRB[5], WRRB[5], WRRRB[5];
	ptype Wl[5], Wr[5], WlN[5], WrN[5], WlS[5], WrS[5], WlF[5], WrF[5], WlB[5], WrB[5];
	int q = 0;	
	if(ix < d_nt_segx-3 && iy < d_nt_segy-3 && iz < d_nt_segz-3 && iy > 2 && iz > 2 && ix > 1)
	{
		q = 0;
		WLLL[q] = d_W0[Id(iy,ix-2,iz)];
		WLL[q]  = d_W0[Id(iy,ix-1,iz)];
		WL[q]   = d_W0[Id(iy,ix,iz)];
		WR[q]   = d_W0[Id(iy,ix+1,iz)];
		WRR[q]  = d_W0[Id(iy,ix+2,iz)];
		WRRR[q] = d_W0[Id(iy,ix+3,iz)];
		
		WLLLN[q] = d_W0[Id(iy+1,ix-2,iz)];
		WLLN[q]  = d_W0[Id(iy+1,ix-1,iz)];
		WLN[q]   = d_W0[Id(iy+1,ix,iz)];
		WRN[q]   = d_W0[Id(iy+1,ix+1,iz)];
		WRRN[q]  = d_W0[Id(iy+1,ix+2,iz)];
		WRRRN[q] = d_W0[Id(iy+1,ix+3,iz)];
		
		WLLLS[q] = d_W0[Id(iy-1,ix-2,iz)];
		WLLS[q]  = d_W0[Id(iy-1,ix-1,iz)];
		WLS[q]   = d_W0[Id(iy-1,ix,iz)];
		WRS[q]   = d_W0[Id(iy-1,ix+1,iz)];
		WRRS[q]  = d_W0[Id(iy-1,ix+2,iz)];
		WRRRS[q] = d_W0[Id(iy-1,ix+3,iz)];

		WLLLF[q] = d_W0[Id(iy,ix-2,iz+1)];
		WLLF[q]  = d_W0[Id(iy,ix-1,iz+1)];
		WLF[q]   = d_W0[Id(iy,ix,iz+1)];
		WRF[q]   = d_W0[Id(iy,ix+1,iz+1)];
		WRRF[q]  = d_W0[Id(iy,ix+2,iz+1)];
		WRRRF[q] = d_W0[Id(iy,ix+3,iz+1)];

		WLLLB[q] = d_W0[Id(iy,ix-2,iz-1)];
		WLLB[q]  = d_W0[Id(iy,ix-1,iz-1)];
		WLB[q]   = d_W0[Id(iy,ix,iz-1)];
		WRB[q]   = d_W0[Id(iy,ix+1,iz-1)];
		WRRB[q]  = d_W0[Id(iy,ix+2,iz-1)];
		WRRRB[q] = d_W0[Id(iy,ix+3,iz-1)];
///////////////////////////////////////////
		q = 1;
		WLLL[q] = d_W1[Id(iy,ix-2,iz)];
		WLL[q]  = d_W1[Id(iy,ix-1,iz)];
		WL[q]   = d_W1[Id(iy,ix,iz)];
		WR[q]   = d_W1[Id(iy,ix+1,iz)];
		WRR[q]  = d_W1[Id(iy,ix+2,iz)];
		WRRR[q] = d_W1[Id(iy,ix+3,iz)];
		
		WLLLN[q] = d_W1[Id(iy+1,ix-2,iz)];
		WLLN[q]  = d_W1[Id(iy+1,ix-1,iz)];
		WLN[q]   = d_W1[Id(iy+1,ix,iz)];
		WRN[q]   = d_W1[Id(iy+1,ix+1,iz)];
		WRRN[q]  = d_W1[Id(iy+1,ix+2,iz)];
		WRRRN[q] = d_W1[Id(iy+1,ix+3,iz)];
		
		WLLLS[q] = d_W1[Id(iy-1,ix-2,iz)];
		WLLS[q]  = d_W1[Id(iy-1,ix-1,iz)];
		WLS[q]   = d_W1[Id(iy-1,ix,iz)];
		WRS[q]   = d_W1[Id(iy-1,ix+1,iz)];
		WRRS[q]  = d_W1[Id(iy-1,ix+2,iz)];
		WRRRS[q] = d_W1[Id(iy-1,ix+3,iz)];

		WLLLF[q] = d_W1[Id(iy,ix-2,iz+1)];
		WLLF[q]  = d_W1[Id(iy,ix-1,iz+1)];
		WLF[q]   = d_W1[Id(iy,ix,iz+1)];
		WRF[q]   = d_W1[Id(iy,ix+1,iz+1)];
		WRRF[q]  = d_W1[Id(iy,ix+2,iz+1)];
		WRRRF[q] = d_W1[Id(iy,ix+3,iz+1)];

		WLLLB[q] = d_W1[Id(iy,ix-2,iz-1)];
		WLLB[q]  = d_W1[Id(iy,ix-1,iz-1)];
		WLB[q]   = d_W1[Id(iy,ix,iz-1)];
		WRB[q]   = d_W1[Id(iy,ix+1,iz-1)];
		WRRB[q]  = d_W1[Id(iy,ix+2,iz-1)];
		WRRRB[q] = d_W1[Id(iy,ix+3,iz-1)];

///////////////////////////////////////////
		q = 2;
		WLLL[q] = d_W2[Id(iy,ix-2,iz)];
		WLL[q]  = d_W2[Id(iy,ix-1,iz)];
		WL[q]   = d_W2[Id(iy,ix,iz)];
		WR[q]   = d_W2[Id(iy,ix+1,iz)];
		WRR[q]  = d_W2[Id(iy,ix+2,iz)];
		WRRR[q] = d_W2[Id(iy,ix+3,iz)];
		
		WLLLN[q] = d_W2[Id(iy+1,ix-2,iz)];
		WLLN[q]  = d_W2[Id(iy+1,ix-1,iz)];
		WLN[q]   = d_W2[Id(iy+1,ix,iz)];
		WRN[q]   = d_W2[Id(iy+1,ix+1,iz)];
		WRRN[q]  = d_W2[Id(iy+1,ix+2,iz)];
		WRRRN[q] = d_W2[Id(iy+1,ix+3,iz)];
		
		WLLLS[q] = d_W2[Id(iy-1,ix-2,iz)];
		WLLS[q]  = d_W2[Id(iy-1,ix-1,iz)];
		WLS[q]   = d_W2[Id(iy-1,ix,iz)];
		WRS[q]   = d_W2[Id(iy-1,ix+1,iz)];
		WRRS[q]  = d_W2[Id(iy-1,ix+2,iz)];
		WRRRS[q] = d_W2[Id(iy-1,ix+3,iz)];

		WLLLF[q] = d_W2[Id(iy,ix-2,iz+1)];
		WLLF[q]  = d_W2[Id(iy,ix-1,iz+1)];
		WLF[q]   = d_W2[Id(iy,ix,iz+1)];
		WRF[q]   = d_W2[Id(iy,ix+1,iz+1)];
		WRRF[q]  = d_W2[Id(iy,ix+2,iz+1)];
		WRRRF[q] = d_W2[Id(iy,ix+3,iz+1)];

		WLLLB[q] = d_W2[Id(iy,ix-2,iz-1)];
		WLLB[q]  = d_W2[Id(iy,ix-1,iz-1)];
		WLB[q]   = d_W2[Id(iy,ix,iz-1)];
		WRB[q]   = d_W2[Id(iy,ix+1,iz-1)];
		WRRB[q]  = d_W2[Id(iy,ix+2,iz-1)];
		WRRRB[q] = d_W2[Id(iy,ix+3,iz-1)];		

///////////////////////////////////////////
		q = 3;
		WLLL[q] = d_W3[Id(iy,ix-2,iz)];
		WLL[q]  = d_W3[Id(iy,ix-1,iz)];
		WL[q]   = d_W3[Id(iy,ix,iz)];
		WR[q]   = d_W3[Id(iy,ix+1,iz)];
		WRR[q]  = d_W3[Id(iy,ix+2,iz)];
		WRRR[q] = d_W3[Id(iy,ix+3,iz)];
	
		WLLLN[q] = d_W3[Id(iy+1,ix-2,iz)];
		WLLN[q]  = d_W3[Id(iy+1,ix-1,iz)];
		WLN[q]   = d_W3[Id(iy+1,ix,iz)];
		WRN[q]   = d_W3[Id(iy+1,ix+1,iz)];
		WRRN[q]  = d_W3[Id(iy+1,ix+2,iz)];
		WRRRN[q] = d_W3[Id(iy+1,ix+3,iz)];

		WLLLS[q] = d_W3[Id(iy-1,ix-2,iz)];
		WLLS[q]  = d_W3[Id(iy-1,ix-1,iz)];
		WLS[q]   = d_W3[Id(iy-1,ix,iz)];
		WRS[q]   = d_W3[Id(iy-1,ix+1,iz)];
		WRRS[q]  = d_W3[Id(iy-1,ix+2,iz)];
		WRRRS[q] = d_W3[Id(iy-1,ix+3,iz)];

		WLLLF[q] = d_W3[Id(iy,ix-2,iz+1)];
		WLLF[q]  = d_W3[Id(iy,ix-1,iz+1)];
		WLF[q]   = d_W3[Id(iy,ix,iz+1)];
		WRF[q]   = d_W3[Id(iy,ix+1,iz+1)];
		WRRF[q]  = d_W3[Id(iy,ix+2,iz+1)];
		WRRRF[q] = d_W3[Id(iy,ix+3,iz+1)];

		WLLLB[q] = d_W1[Id(iy,ix-2,iz-1)];
		WLLB[q]  = d_W1[Id(iy,ix-1,iz-1)];
		WLB[q]   = d_W1[Id(iy,ix,iz-1)];
		WRB[q]   = d_W1[Id(iy,ix+1,iz-1)];
		WRRB[q]  = d_W1[Id(iy,ix+2,iz-1)];
		WRRRB[q] = d_W1[Id(iy,ix+3,iz-1)];

///////////////////////////////////////////
		q = 4;
		WLLL[q] = d_W4[Id(iy,ix-2,iz)];
		WLL[q]  = d_W4[Id(iy,ix-1,iz)];
		WL[q]   = d_W4[Id(iy,ix,iz)];
		WR[q]   = d_W4[Id(iy,ix+1,iz)];
		WRR[q]  = d_W4[Id(iy,ix+2,iz)];
		WRRR[q] = d_W4[Id(iy,ix+3,iz)];
		
		WLLLN[q] = d_W4[Id(iy+1,ix-2,iz)];
		WLLN[q]  = d_W4[Id(iy+1,ix-1,iz)];
		WLN[q]   = d_W4[Id(iy+1,ix,iz)];
		WRN[q]   = d_W4[Id(iy+1,ix+1,iz)];
		WRRN[q]  = d_W4[Id(iy+1,ix+2,iz)];
		WRRRN[q] = d_W4[Id(iy+1,ix+3,iz)];
		
		WLLLS[q] = d_W4[Id(iy-1,ix-2,iz)];
		WLLS[q]  = d_W4[Id(iy-1,ix-1,iz)];
		WLS[q]   = d_W4[Id(iy-1,ix,iz)];
		WRS[q]   = d_W4[Id(iy-1,ix+1,iz)];
		WRRS[q]  = d_W4[Id(iy-1,ix+2,iz)];
		WRRRS[q] = d_W4[Id(iy-1,ix+3,iz)];

		WLLLF[q] = d_W4[Id(iy,ix-2,iz+1)];
		WLLF[q]  = d_W4[Id(iy,ix-1,iz+1)];
		WLF[q]   = d_W4[Id(iy,ix,iz+1)];
		WRF[q]   = d_W4[Id(iy,ix+1,iz+1)];
		WRRF[q]  = d_W4[Id(iy,ix+2,iz+1)];
		WRRRF[q] = d_W4[Id(iy,ix+3,iz+1)];

		WLLLB[q] = d_W4[Id(iy,ix-2,iz-1)];
		WLLB[q]  = d_W4[Id(iy,ix-1,iz-1)];
		WLB[q]   = d_W4[Id(iy,ix,iz-1)];
		WRB[q]   = d_W4[Id(iy,ix+1,iz-1)];
		WRRB[q]  = d_W4[Id(iy,ix+2,iz-1)];
		WRRRB[q] = d_W4[Id(iy,ix+3,iz-1)];		
		
		d_ApplyWENO(WLLL, WLL, WL, WR, WRR, WRRR, Wl, Wr);
		d_ApplyWENO(WLLLN, WLLN, WLN, WRN, WRRN, WRRRN, WlN, WrN);
		d_ApplyWENO(WLLLS, WLLS, WLS, WRS, WRRS, WRRRS, WlS, WrS);
		d_ApplyWENO(WLLLF, WLLF, WLF, WRF, WRRF, WRRRF, WlF, WrF);
		d_ApplyWENO(WLLLB, WLLB, WLB, WRB, WRRB, WRRRB, WlB, WrB);
		
		d_Wl0[Id(iy,ix,iz)] = Wl[0];
		d_Wl1[Id(iy,ix,iz)] = Wl[1];
		d_Wl2[Id(iy,ix,iz)] = Wl[2];
		d_Wl3[Id(iy,ix,iz)] = Wl[3];
		d_Wl4[Id(iy,ix,iz)] = Wl[4];
		
		d_Wr0[Id(iy,ix,iz)] = Wr[0];
		d_Wr1[Id(iy,ix,iz)] = Wr[1];
		d_Wr2[Id(iy,ix,iz)] = Wr[2];
		d_Wr3[Id(iy,ix,iz)] = Wr[3];
		d_Wr4[Id(iy,ix,iz)] = Wr[4];
		
		d_DW0xl[Id(iy,ix,iz)] = 2*(Wl[0] - WL[0]) / d_dx;
		d_DW1xl[Id(iy,ix,iz)] = 2*(Wl[1] - WL[1]) / d_dx;
		d_DW2xl[Id(iy,ix,iz)] = 2*(Wl[2] - WL[2]) / d_dx;
		d_DW3xl[Id(iy,ix,iz)] = 2*(Wl[3] - WL[3]) / d_dx;
		d_DW4xl[Id(iy,ix,iz)] = 2*(Wl[4] - WL[4]) / d_dx;
	
		d_DW0xr[Id(iy,ix,iz)] = 2*(WR[0] - Wr[0]) / d_dx;
		d_DW1xr[Id(iy,ix,iz)] = 2*(WR[1] - Wr[1]) / d_dx;
		d_DW2xr[Id(iy,ix,iz)] = 2*(WR[2] - Wr[2]) / d_dx;
		d_DW3xr[Id(iy,ix,iz)] = 2*(WR[3] - Wr[3]) / d_dx;
		d_DW4xr[Id(iy,ix,iz)] = 2*(WR[4] - Wr[4]) / d_dx;
		
		d_DW0yl[Id(iy,ix,iz)] = 0.5*(WlN[0] - WlS[0]) / d_dx;
		d_DW1yl[Id(iy,ix,iz)] = 0.5*(WlN[1] - WlS[1]) / d_dx;
		d_DW2yl[Id(iy,ix,iz)] = 0.5*(WlN[2] - WlS[2]) / d_dx;
		d_DW3yl[Id(iy,ix,iz)] = 0.5*(WlN[3] - WlS[3]) / d_dx;
		d_DW4yl[Id(iy,ix,iz)] = 0.5*(WlN[4] - WlS[4]) / d_dx;
		
		d_DW0yr[Id(iy,ix,iz)] = 0.5*(WrN[0] - WrS[0]) / d_dx;
		d_DW1yr[Id(iy,ix,iz)] = 0.5*(WrN[1] - WrS[1]) / d_dx;
		d_DW2yr[Id(iy,ix,iz)] = 0.5*(WrN[2] - WrS[2]) / d_dx;
		d_DW3yr[Id(iy,ix,iz)] = 0.5*(WrN[3] - WrS[3]) / d_dx;
		d_DW4yr[Id(iy,ix,iz)] = 0.5*(WrN[4] - WrS[4]) / d_dx;

		d_DW0zl[Id(iy,ix,iz)] = 0.5*(WlF[0] - WlB[0]) / d_dx;
		d_DW1zl[Id(iy,ix,iz)] = 0.5*(WlF[1] - WlB[1]) / d_dx;
		d_DW2zl[Id(iy,ix,iz)] = 0.5*(WlF[2] - WlB[2]) / d_dx;
		d_DW3zl[Id(iy,ix,iz)] = 0.5*(WlF[3] - WlB[3]) / d_dx;
		d_DW4zl[Id(iy,ix,iz)] = 0.5*(WlF[4] - WlB[4]) / d_dx;
		
		d_DW0zr[Id(iy,ix,iz)] = 0.5*(WrF[0] - WrB[0]) / d_dx;
		d_DW1zr[Id(iy,ix,iz)] = 0.5*(WrF[1] - WrB[1]) / d_dx;
		d_DW2zr[Id(iy,ix,iz)] = 0.5*(WrF[2] - WrB[2]) / d_dx;
		d_DW3zr[Id(iy,ix,iz)] = 0.5*(WrF[3] - WrB[3]) / d_dx;
		d_DW4zr[Id(iy,ix,iz)] = 0.5*(WrF[4] - WrB[4]) / d_dx;
	}	
}


/**-----------------------------------------------------------------------------------------*
 * Global Function : Calculates Derivatives required for calculation of flux in Y-direction
 *------------------------------------------------------------------------------------------*/

__global__ void derivsY_Kernel(ptype *d_W0, ptype *d_Wl0, ptype *d_Wr0, ptype *d_DW0xl, ptype *d_DW0yl, ptype *d_DW0zl, ptype *d_DW0xr, ptype *d_DW0yr, ptype *d_DW0zr,
							   ptype *d_W1, ptype *d_Wl1, ptype *d_Wr1, ptype *d_DW1xl, ptype *d_DW1yl, ptype *d_DW1zl, ptype *d_DW1xr, ptype *d_DW1yr, ptype *d_DW1zr,
							   ptype *d_W2, ptype *d_Wl2, ptype *d_Wr2, ptype *d_DW2xl, ptype *d_DW2yl, ptype *d_DW2zl, ptype *d_DW2xr, ptype *d_DW2yr, ptype *d_DW2zr,
							   ptype *d_W3, ptype *d_Wl3, ptype *d_Wr3, ptype *d_DW3xl, ptype *d_DW3yl, ptype *d_DW3zl, ptype *d_DW3xr, ptype *d_DW3yr, ptype *d_DW3zr,
							   ptype *d_W4, ptype *d_Wl4, ptype *d_Wr4, ptype *d_DW4xl, ptype *d_DW4yl, ptype *d_DW4zl, ptype *d_DW4xr, ptype *d_DW4yr, ptype *d_DW4zr)
{
	int I = blockIdx.x*(blockDim.x) + threadIdx.x;

	int iy =  I/(d_nt_segx*d_nt_segz);
	int ix =  (I%(d_nt_segx*d_nt_segz))/d_nt_segz;
	int iz =  I%d_nt_segz;

	ptype WLLL[5], WLL[5], WL[5], WR[5], WRR[5], WRRR[5];
	ptype WLLLN[5], WLLN[5], WLN[5], WRN[5], WRRN[5], WRRRN[5];
	ptype WLLLS[5], WLLS[5], WLS[5], WRS[5], WRRS[5], WRRRS[5];
	ptype WLLLF[5], WLLF[5], WLF[5], WRF[5], WRRF[5], WRRRF[5];
	ptype WLLLB[5], WLLB[5], WLB[5], WRB[5], WRRB[5], WRRRB[5];
	ptype Wl[5], Wr[5], WlN[5], WrN[5], WlS[5], WrS[5], WlF[5], WrF[5], WlB[5], WrB[5];
	int q = 0;	
			
	if(ix < d_nt_segx-3 && iy < d_nt_segy-3 && iz < d_nt_segz-3 && iy > 1 && iz > 2 && ix > 2)
	{	
		q = 0;
		WLLL[q] = d_W0[Id(iy-2,ix,iz)];
		WLL[q]  = d_W0[Id(iy-1,ix,iz)];
		WL[q]   = d_W0[Id(iy,ix,iz)];
		WR[q]   = d_W0[Id(iy+1,ix,iz)];
		WRR[q]  = d_W0[Id(iy+2,ix,iz)];
		WRRR[q] = d_W0[Id(iy+3,ix,iz)];
		
		WLLLN[q] = d_W0[Id(iy-2,ix-1,iz)];
		WLLN[q]  = d_W0[Id(iy-1,ix-1,iz)];
		WLN[q]   = d_W0[Id(iy,ix-1,iz)];
		WRN[q]   = d_W0[Id(iy+1,ix-1,iz)];
		WRRN[q]  = d_W0[Id(iy+2,ix-1,iz)];
		WRRRN[q] = d_W0[Id(iy+3,ix-1,iz)];
		
		WLLLS[q] = d_W0[Id(iy-2,ix+1,iz)];
		WLLS[q]  = d_W0[Id(iy-1,ix+1,iz)];
		WLS[q]   = d_W0[Id(iy,ix+1,iz)];
		WRS[q]   = d_W0[Id(iy+1,ix+1,iz)];
		WRRS[q]  = d_W0[Id(iy+2,ix+1,iz)];
		WRRRS[q] = d_W0[Id(iy+3,ix+1,iz)];

		WLLLF[q] = d_W0[Id(iy-2,ix,iz+1)];
		WLLF[q]  = d_W0[Id(iy-1,ix,iz+1)];
		WLF[q]   = d_W0[Id(iy,ix,iz+1)];
		WRF[q]   = d_W0[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W0[Id(iy+2,ix,iz+1)];
		WRRRF[q] = d_W0[Id(iy+3,ix,iz+1)];

		WLLLB[q] = d_W0[Id(iy-2,ix,iz-1)];
		WLLB[q]  = d_W0[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W0[Id(iy,ix,iz-1)];
		WRB[q]   = d_W0[Id(iy+1,ix,iz-1)];
		WRRB[q]  = d_W0[Id(iy+2,ix,iz-1)];
		WRRRB[q] = d_W0[Id(iy+3,ix,iz-1)];
////////////////////////////////////////////
		q = 1;
		WLLL[q] = d_W1[Id(iy-2,ix,iz)];
		WLL[q]  = d_W1[Id(iy-1,ix,iz)];
		WL[q]   = d_W1[Id(iy,ix,iz)];
		WR[q]   = d_W1[Id(iy+1,ix,iz)];
		WRR[q]  = d_W1[Id(iy+2,ix,iz)];
		WRRR[q] = d_W1[Id(iy+3,ix,iz)];
		
		WLLLN[q] = d_W1[Id(iy-2,ix-1,iz)];
		WLLN[q]  = d_W1[Id(iy-1,ix-1,iz)];
		WLN[q]   = d_W1[Id(iy,ix-1,iz)];
		WRN[q]   = d_W1[Id(iy+1,ix-1,iz)];
		WRRN[q]  = d_W1[Id(iy+2,ix-1,iz)];
		WRRRN[q] = d_W1[Id(iy+3,ix-1,iz)];
		
		WLLLS[q] = d_W1[Id(iy-2,ix+1,iz)];
		WLLS[q]  = d_W1[Id(iy-1,ix+1,iz)];
		WLS[q]   = d_W1[Id(iy,ix+1,iz)];
		WRS[q]   = d_W1[Id(iy+1,ix+1,iz)];
		WRRS[q]  = d_W1[Id(iy+2,ix+1,iz)];
		WRRRS[q] = d_W1[Id(iy+3,ix+1,iz)];

		WLLLF[q] = d_W1[Id(iy-2,ix,iz+1)];
		WLLF[q]  = d_W1[Id(iy-1,ix,iz+1)];
		WLF[q]   = d_W1[Id(iy,ix,iz+1)];
		WRF[q]   = d_W1[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W1[Id(iy+2,ix,iz+1)];
		WRRRF[q] = d_W1[Id(iy+3,ix,iz+1)];

		WLLLB[q] = d_W1[Id(iy-2,ix,iz-1)];
		WLLB[q]  = d_W1[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W1[Id(iy,ix,iz-1)];
		WRB[q]   = d_W1[Id(iy+1,ix,iz-1)];
		WRRB[q]  = d_W1[Id(iy+2,ix,iz-1)];
		WRRRB[q] = d_W1[Id(iy+3,ix,iz-1)];
////////////////////////////////////////////
		q = 2;
		WLLL[q] = d_W2[Id(iy-2,ix,iz)];
		WLL[q]  = d_W2[Id(iy-1,ix,iz)];
		WL[q]   = d_W2[Id(iy,ix,iz)];
		WR[q]   = d_W2[Id(iy+1,ix,iz)];
		WRR[q]  = d_W2[Id(iy+2,ix,iz)];
		WRRR[q] = d_W2[Id(iy+3,ix,iz)];
		
		WLLLN[q] = d_W2[Id(iy-2,ix-1,iz)];
		WLLN[q]  = d_W2[Id(iy-1,ix-1,iz)];
		WLN[q]   = d_W2[Id(iy,ix-1,iz)];
		WRN[q]   = d_W2[Id(iy+1,ix-1,iz)];
		WRRN[q]  = d_W2[Id(iy+2,ix-1,iz)];
		WRRRN[q] = d_W2[Id(iy+3,ix-1,iz)];
		
		WLLLS[q] = d_W2[Id(iy-2,ix+1,iz)];
		WLLS[q]  = d_W2[Id(iy-1,ix+1,iz)];
		WLS[q]   = d_W2[Id(iy,ix+1,iz)];
		WRS[q]   = d_W2[Id(iy+1,ix+1,iz)];
		WRRS[q]  = d_W2[Id(iy+2,ix+1,iz)];
		WRRRS[q] = d_W2[Id(iy+3,ix+1,iz)];

		WLLLF[q] = d_W2[Id(iy-2,ix,iz+1)];
		WLLF[q]  = d_W2[Id(iy-1,ix,iz+1)];
		WLF[q]   = d_W2[Id(iy,ix,iz+1)];
		WRF[q]   = d_W2[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W2[Id(iy+2,ix,iz+1)];
		WRRRF[q] = d_W2[Id(iy+3,ix,iz+1)];

		WLLLB[q] = d_W2[Id(iy-2,ix,iz-1)];
		WLLB[q]  = d_W2[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W2[Id(iy,ix,iz-1)];
		WRB[q]   = d_W2[Id(iy+1,ix,iz-1)];
		WRRB[q]  = d_W2[Id(iy+2,ix,iz-1)];
		WRRRB[q] = d_W2[Id(iy+3,ix,iz-1)];
////////////////////////////////////////////
		q = 3;
		WLLL[q] = d_W3[Id(iy-2,ix,iz)];
		WLL[q]  = d_W3[Id(iy-1,ix,iz)];
		WL[q]   = d_W3[Id(iy,ix,iz)];
		WR[q]   = d_W3[Id(iy+1,ix,iz)];
		WRR[q]  = d_W3[Id(iy+2,ix,iz)];
		WRRR[q] = d_W3[Id(iy+3,ix,iz)];
		
		WLLLN[q] = d_W3[Id(iy-2,ix-1,iz)];
		WLLN[q]  = d_W3[Id(iy-1,ix-1,iz)];
		WLN[q]   = d_W3[Id(iy,ix-1,iz)];
		WRN[q]   = d_W3[Id(iy+1,ix-1,iz)];
		WRRN[q]  = d_W3[Id(iy+2,ix-1,iz)];
		WRRRN[q] = d_W3[Id(iy+3,ix-1,iz)];
		
		WLLLS[q] = d_W3[Id(iy-2,ix+1,iz)];
		WLLS[q]  = d_W3[Id(iy-1,ix+1,iz)];
		WLS[q]   = d_W3[Id(iy,ix+1,iz)];
		WRS[q]   = d_W3[Id(iy+1,ix+1,iz)];
		WRRS[q]  = d_W3[Id(iy+2,ix+1,iz)];
		WRRRS[q] = d_W3[Id(iy+3,ix+1,iz)];

		WLLLF[q] = d_W3[Id(iy-2,ix,iz+1)];
		WLLF[q]  = d_W3[Id(iy-1,ix,iz+1)];
		WLF[q]   = d_W3[Id(iy,ix,iz+1)];
		WRF[q]   = d_W3[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W3[Id(iy+2,ix,iz+1)];
		WRRRF[q] = d_W3[Id(iy+3,ix,iz+1)];

		WLLLB[q] = d_W3[Id(iy-2,ix,iz-1)];
		WLLB[q]  = d_W3[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W3[Id(iy,ix,iz-1)];
		WRB[q]   = d_W3[Id(iy+1,ix,iz-1)];
		WRRB[q]  = d_W3[Id(iy+2,ix,iz-1)];
		WRRRB[q] = d_W3[Id(iy+3,ix,iz-1)];
////////////////////////////////////////////
		q = 4;
		WLLL[q] = d_W4[Id(iy-2,ix,iz)];
		WLL[q]  = d_W4[Id(iy-1,ix,iz)];
		WL[q]   = d_W4[Id(iy,ix,iz)];
		WR[q]   = d_W4[Id(iy+1,ix,iz)];
		WRR[q]  = d_W4[Id(iy+2,ix,iz)];
		WRRR[q] = d_W4[Id(iy+3,ix,iz)];
		
		WLLLN[q] = d_W4[Id(iy-2,ix-1,iz)];
		WLLN[q]  = d_W4[Id(iy-1,ix-1,iz)];
		WLN[q]   = d_W4[Id(iy,ix-1,iz)];
		WRN[q]   = d_W4[Id(iy+1,ix-1,iz)];
		WRRN[q]  = d_W4[Id(iy+2,ix-1,iz)];
		WRRRN[q] = d_W4[Id(iy+3,ix-1,iz)];
		
		WLLLS[q] = d_W4[Id(iy-2,ix+1,iz)];
		WLLS[q]  = d_W4[Id(iy-1,ix+1,iz)];
		WLS[q]   = d_W4[Id(iy,ix+1,iz)];
		WRS[q]   = d_W4[Id(iy+1,ix+1,iz)];
		WRRS[q]  = d_W4[Id(iy+2,ix+1,iz)];
		WRRRS[q] = d_W4[Id(iy+3,ix+1,iz)];

		WLLLF[q] = d_W4[Id(iy-2,ix,iz+1)];
		WLLF[q]  = d_W4[Id(iy-1,ix,iz+1)];
		WLF[q]   = d_W4[Id(iy,ix,iz+1)];
		WRF[q]   = d_W4[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W4[Id(iy+2,ix,iz+1)];
		WRRRF[q] = d_W4[Id(iy+3,ix,iz+1)];

		WLLLB[q] = d_W4[Id(iy-2,ix,iz-1)];
		WLLB[q]  = d_W4[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W4[Id(iy,ix,iz-1)];
		WRB[q]   = d_W4[Id(iy+1,ix,iz-1)];
		WRRB[q]  = d_W4[Id(iy+2,ix,iz-1)];
		WRRRB[q] = d_W4[Id(iy+3,ix,iz-1)];
		
////////////////////////////////////////////

		d_ApplyWENO(WLLL, WLL, WL, WR, WRR, WRRR, Wl, Wr);
		d_ApplyWENO(WLLLN, WLLN, WLN, WRN, WRRN, WRRRN, WlN, WrN);
		d_ApplyWENO(WLLLS, WLLS, WLS, WRS, WRRS, WRRRS, WlS, WrS);
		d_ApplyWENO(WLLLF, WLLF, WLF, WRF, WRRF, WRRRF, WlF, WrF);
		d_ApplyWENO(WLLLB, WLLB, WLB, WRB, WRRB, WRRRB, WlB, WrB);
		
		d_Wl0[Id(iy,ix,iz)] = Wl[0];
		d_Wl1[Id(iy,ix,iz)] = Wl[1];
		d_Wl2[Id(iy,ix,iz)] = Wl[2];
		d_Wl3[Id(iy,ix,iz)] = Wl[3];
		d_Wl4[Id(iy,ix,iz)] = Wl[4];
		
		d_Wr0[Id(iy,ix,iz)] = Wr[0];
		d_Wr1[Id(iy,ix,iz)] = Wr[1];
		d_Wr2[Id(iy,ix,iz)] = Wr[2];
		d_Wr3[Id(iy,ix,iz)] = Wr[3];
		d_Wr4[Id(iy,ix,iz)] = Wr[4];
		
		d_DW0xl[Id(iy,ix,iz)] = 2*(Wl[0] - WL[0]) / d_dx;
		d_DW1xl[Id(iy,ix,iz)] = 2*(Wl[1] - WL[1]) / d_dx;
		d_DW2xl[Id(iy,ix,iz)] = 2*(Wl[2] - WL[2]) / d_dx;
		d_DW3xl[Id(iy,ix,iz)] = 2*(Wl[3] - WL[3]) / d_dx;
		d_DW4xl[Id(iy,ix,iz)] = 2*(Wl[4] - WL[4]) / d_dx;
	
		d_DW0xr[Id(iy,ix,iz)] = 2*(WR[0] - Wr[0]) / d_dx;
		d_DW1xr[Id(iy,ix,iz)] = 2*(WR[1] - Wr[1]) / d_dx;
		d_DW2xr[Id(iy,ix,iz)] = 2*(WR[2] - Wr[2]) / d_dx;
		d_DW3xr[Id(iy,ix,iz)] = 2*(WR[3] - Wr[3]) / d_dx;
		d_DW4xr[Id(iy,ix,iz)] = 2*(WR[4] - Wr[4]) / d_dx;
		
		d_DW0yl[Id(iy,ix,iz)] = -0.5*(WlN[0] - WlS[0]) / d_dx;
		d_DW1yl[Id(iy,ix,iz)] = -0.5*(WlN[1] - WlS[1]) / d_dx;
		d_DW2yl[Id(iy,ix,iz)] = -0.5*(WlN[2] - WlS[2]) / d_dx;
		d_DW3yl[Id(iy,ix,iz)] = -0.5*(WlN[3] - WlS[3]) / d_dx;
		d_DW4yl[Id(iy,ix,iz)] = -0.5*(WlN[4] - WlS[4]) / d_dx;
		
		d_DW0yr[Id(iy,ix,iz)] = -0.5*(WrN[0] - WrS[0]) / d_dx;
		d_DW1yr[Id(iy,ix,iz)] = -0.5*(WrN[1] - WrS[1]) / d_dx;
		d_DW2yr[Id(iy,ix,iz)] = -0.5*(WrN[2] - WrS[2]) / d_dx;
		d_DW3yr[Id(iy,ix,iz)] = -0.5*(WrN[3] - WrS[3]) / d_dx;
		d_DW4yr[Id(iy,ix,iz)] = -0.5*(WrN[4] - WrS[4]) / d_dx;

		d_DW0zl[Id(iy,ix,iz)] = 0.5*(WlF[0] - WlB[0]) / d_dx;
		d_DW1zl[Id(iy,ix,iz)] = 0.5*(WlF[1] - WlB[1]) / d_dx;
		d_DW2zl[Id(iy,ix,iz)] = 0.5*(WlF[2] - WlB[2]) / d_dx;
		d_DW3zl[Id(iy,ix,iz)] = 0.5*(WlF[3] - WlB[3]) / d_dx;
		d_DW4zl[Id(iy,ix,iz)] = 0.5*(WlF[4] - WlB[4]) / d_dx;
		
		d_DW0zr[Id(iy,ix,iz)] = 0.5*(WrF[0] - WrB[0]) / d_dx;
		d_DW1zr[Id(iy,ix,iz)] = 0.5*(WrF[1] - WrB[1]) / d_dx;
		d_DW2zr[Id(iy,ix,iz)] = 0.5*(WrF[2] - WrB[2]) / d_dx;
		d_DW3zr[Id(iy,ix,iz)] = 0.5*(WrF[3] - WrB[3]) / d_dx;
		d_DW4zr[Id(iy,ix,iz)] = 0.5*(WrF[4] - WrB[4]) / d_dx;
	}	
}	


/**-----------------------------------------------------------------------------------------*
 * Global Function : Calculates Derivatives required for calculation of flux in Z-direction
 *------------------------------------------------------------------------------------------*/
__global__ void derivsZ_Kernel(ptype *d_W0, ptype *d_Wl0, ptype *d_Wr0, ptype *d_DW0xl, ptype *d_DW0yl, ptype *d_DW0zl, ptype *d_DW0xr, ptype *d_DW0yr, ptype *d_DW0zr,
							   ptype *d_W1, ptype *d_Wl1, ptype *d_Wr1, ptype *d_DW1xl, ptype *d_DW1yl, ptype *d_DW1zl, ptype *d_DW1xr, ptype *d_DW1yr, ptype *d_DW1zr,
							   ptype *d_W2, ptype *d_Wl2, ptype *d_Wr2, ptype *d_DW2xl, ptype *d_DW2yl, ptype *d_DW2zl, ptype *d_DW2xr, ptype *d_DW2yr, ptype *d_DW2zr,
							   ptype *d_W3, ptype *d_Wl3, ptype *d_Wr3, ptype *d_DW3xl, ptype *d_DW3yl, ptype *d_DW3zl, ptype *d_DW3xr, ptype *d_DW3yr, ptype *d_DW3zr,
							   ptype *d_W4, ptype *d_Wl4, ptype *d_Wr4, ptype *d_DW4xl, ptype *d_DW4yl, ptype *d_DW4zl, ptype *d_DW4xr, ptype *d_DW4yr, ptype *d_DW4zr)
{
	int I = blockIdx.x*(blockDim.x) + threadIdx.x;

	int iy =  I/(d_nt_segx*d_nt_segz);
	int ix =  (I%(d_nt_segx*d_nt_segz))/d_nt_segz;
	int iz =  I%d_nt_segz;

	ptype WLLL[5], WLL[5], WL[5], WR[5], WRR[5], WRRR[5];
	ptype WLLLN[5], WLLN[5], WLN[5], WRN[5], WRRN[5], WRRRN[5];
	ptype WLLLS[5], WLLS[5], WLS[5], WRS[5], WRRS[5], WRRRS[5];
	ptype WLLLF[5], WLLF[5], WLF[5], WRF[5], WRRF[5], WRRRF[5];
	ptype WLLLB[5], WLLB[5], WLB[5], WRB[5], WRRB[5], WRRRB[5];
	ptype Wl[5], Wr[5], WlN[5], WrN[5], WlS[5], WrS[5], WlF[5], WrF[5], WlB[5], WrB[5];
	int q = 0;	
			
	if(ix < d_nt_segx-3 && iy < d_nt_segy-3 && iz < d_nt_segz-3 && iy > 2 && iz > 1 && ix > 2)
	{	
		q = 0;
		WLLL[q] = d_W0[Id(iy,ix,iz-2)];
		WLL[q]  = d_W0[Id(iy,ix,iz-1)];
		WL[q]   = d_W0[Id(iy,ix,iz)];
		WR[q]   = d_W0[Id(iy,ix,iz+1)];
		WRR[q]  = d_W0[Id(iy,ix,iz+2)];
		WRRR[q] = d_W0[Id(iy,ix,iz+3)];
		
		WLLLN[q] = d_W0[Id(iy,ix+1,iz-2)];
		WLLN[q]  = d_W0[Id(iy,ix+1,iz-1)];
		WLN[q]   = d_W0[Id(iy,ix+1,iz)];
		WRN[q]   = d_W0[Id(iy,ix+1,iz+1)];
		WRRN[q]  = d_W0[Id(iy,ix+1,iz+2)];
		WRRRN[q] = d_W0[Id(iy,ix+1,iz+3)];
		
		WLLLS[q] = d_W0[Id(iy,ix-1,iz-2)];
		WLLS[q]  = d_W0[Id(iy,ix-1,iz-1)];
		WLS[q]   = d_W0[Id(iy,ix-1,iz)];
		WRS[q]   = d_W0[Id(iy,ix-1,iz+1)];
		WRRS[q]  = d_W0[Id(iy,ix-1,iz+2)];
		WRRRS[q] = d_W0[Id(iy,ix-1,iz+3)];

		WLLLF[q] = d_W0[Id(iy+1,ix,iz-2)];
		WLLF[q]  = d_W0[Id(iy+1,ix,iz-1)];
		WLF[q]   = d_W0[Id(iy+1,ix,iz)];
		WRF[q]   = d_W0[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W0[Id(iy+1,ix,iz+2)];
		WRRRF[q] = d_W0[Id(iy+1,ix,iz+3)];

		WLLLB[q] = d_W0[Id(iy-1,ix,iz-2)];
		WLLB[q]  = d_W0[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W0[Id(iy-1,ix,iz)];
		WRB[q]   = d_W0[Id(iy-1,ix,iz+1)];
		WRRB[q]  = d_W0[Id(iy-1,ix,iz+2)];
		WRRRB[q] = d_W0[Id(iy-1,ix,iz+3)];
///////////////////////////////////////////////////////
		q = 1;
		WLLL[q] = d_W1[Id(iy,ix,iz-2)];
		WLL[q]  = d_W1[Id(iy,ix,iz-1)];
		WL[q]   = d_W1[Id(iy,ix,iz)];
		WR[q]   = d_W1[Id(iy,ix,iz+1)];
		WRR[q]  = d_W1[Id(iy,ix,iz+2)];
		WRRR[q] = d_W1[Id(iy,ix,iz+3)];
		
		WLLLN[q] = d_W1[Id(iy,ix+1,iz-2)];
		WLLN[q]  = d_W1[Id(iy,ix+1,iz-1)];
		WLN[q]   = d_W1[Id(iy,ix+1,iz)];
		WRN[q]   = d_W1[Id(iy,ix+1,iz+1)];
		WRRN[q]  = d_W1[Id(iy,ix+1,iz+2)];
		WRRRN[q] = d_W1[Id(iy,ix+1,iz+3)];
		
		WLLLS[q] = d_W1[Id(iy,ix-1,iz-2)];
		WLLS[q]  = d_W1[Id(iy,ix-1,iz-1)];
		WLS[q]   = d_W1[Id(iy,ix-1,iz)];
		WRS[q]   = d_W1[Id(iy,ix-1,iz+1)];
		WRRS[q]  = d_W1[Id(iy,ix-1,iz+2)];
		WRRRS[q] = d_W1[Id(iy,ix-1,iz+3)];

		WLLLF[q] = d_W1[Id(iy+1,ix,iz-2)];
		WLLF[q]  = d_W1[Id(iy+1,ix,iz-1)];
		WLF[q]   = d_W1[Id(iy+1,ix,iz)];
		WRF[q]   = d_W1[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W1[Id(iy+1,ix,iz+2)];
		WRRRF[q] = d_W1[Id(iy+1,ix,iz+3)];

		WLLLB[q] = d_W1[Id(iy-1,ix,iz-2)];
		WLLB[q]  = d_W1[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W1[Id(iy-1,ix,iz)];
		WRB[q]   = d_W1[Id(iy-1,ix,iz+1)];
		WRRB[q]  = d_W1[Id(iy-1,ix,iz+2)];
		WRRRB[q] = d_W1[Id(iy-1,ix,iz+3)];
///////////////////////////////////////////////////////
		q = 2;
		WLLL[q] = d_W2[Id(iy,ix,iz-2)];
		WLL[q]  = d_W2[Id(iy,ix,iz-1)];
		WL[q]   = d_W2[Id(iy,ix,iz)];
		WR[q]   = d_W2[Id(iy,ix,iz+1)];
		WRR[q]  = d_W2[Id(iy,ix,iz+2)];
		WRRR[q] = d_W2[Id(iy,ix,iz+3)];
		
		WLLLN[q] = d_W2[Id(iy,ix+1,iz-2)];
		WLLN[q]  = d_W2[Id(iy,ix+1,iz-1)];
		WLN[q]   = d_W2[Id(iy,ix+1,iz)];
		WRN[q]   = d_W2[Id(iy,ix+1,iz+1)];
		WRRN[q]  = d_W2[Id(iy,ix+1,iz+2)];
		WRRRN[q] = d_W2[Id(iy,ix+1,iz+3)];
		
		WLLLS[q] = d_W2[Id(iy,ix-1,iz-2)];
		WLLS[q]  = d_W2[Id(iy,ix-1,iz-1)];
		WLS[q]   = d_W2[Id(iy,ix-1,iz)];
		WRS[q]   = d_W2[Id(iy,ix-1,iz+1)];
		WRRS[q]  = d_W2[Id(iy,ix-1,iz+2)];
		WRRRS[q] = d_W2[Id(iy,ix-1,iz+3)];

		WLLLF[q] = d_W2[Id(iy+1,ix,iz-2)];
		WLLF[q]  = d_W2[Id(iy+1,ix,iz-1)];
		WLF[q]   = d_W2[Id(iy+1,ix,iz)];
		WRF[q]   = d_W2[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W2[Id(iy+1,ix,iz+2)];
		WRRRF[q] = d_W2[Id(iy+1,ix,iz+3)];

		WLLLB[q] = d_W2[Id(iy-1,ix,iz-2)];
		WLLB[q]  = d_W2[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W2[Id(iy-1,ix,iz)];
		WRB[q]   = d_W2[Id(iy-1,ix,iz+1)];
		WRRB[q]  = d_W2[Id(iy-1,ix,iz+2)];
		WRRRB[q] = d_W2[Id(iy-1,ix,iz+3)];
///////////////////////////////////////////////////////
		q = 3;
		WLLL[q] = d_W3[Id(iy,ix,iz-2)];
		WLL[q]  = d_W3[Id(iy,ix,iz-1)];
		WL[q]   = d_W3[Id(iy,ix,iz)];
		WR[q]   = d_W3[Id(iy,ix,iz+1)];
		WRR[q]  = d_W3[Id(iy,ix,iz+2)];
		WRRR[q] = d_W3[Id(iy,ix,iz+3)];
		
		WLLLN[q] = d_W3[Id(iy,ix+1,iz-2)];
		WLLN[q]  = d_W3[Id(iy,ix+1,iz-1)];
		WLN[q]   = d_W3[Id(iy,ix+1,iz)];
		WRN[q]   = d_W3[Id(iy,ix+1,iz+1)];
		WRRN[q]  = d_W3[Id(iy,ix+1,iz+2)];
		WRRRN[q] = d_W3[Id(iy,ix+1,iz+3)];
		
		WLLLS[q] = d_W3[Id(iy,ix-1,iz-2)];
		WLLS[q]  = d_W3[Id(iy,ix-1,iz-1)];
		WLS[q]   = d_W3[Id(iy,ix-1,iz)];
		WRS[q]   = d_W3[Id(iy,ix-1,iz+1)];
		WRRS[q]  = d_W3[Id(iy,ix-1,iz+2)];
		WRRRS[q] = d_W3[Id(iy,ix-1,iz+3)];

		WLLLF[q] = d_W3[Id(iy+1,ix,iz-2)];
		WLLF[q]  = d_W3[Id(iy+1,ix,iz-1)];
		WLF[q]   = d_W3[Id(iy+1,ix,iz)];
		WRF[q]   = d_W3[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W3[Id(iy+1,ix,iz+2)];
		WRRRF[q] = d_W3[Id(iy+1,ix,iz+3)];

		WLLLB[q] = d_W3[Id(iy-1,ix,iz-2)];
		WLLB[q]  = d_W3[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W3[Id(iy-1,ix,iz)];
		WRB[q]   = d_W3[Id(iy-1,ix,iz+1)];
		WRRB[q]  = d_W3[Id(iy-1,ix,iz+2)];
		WRRRB[q] = d_W3[Id(iy-1,ix,iz+3)];
///////////////////////////////////////////////////////
		q = 4;
		WLLL[q] = d_W4[Id(iy,ix,iz-2)];
		WLL[q]  = d_W4[Id(iy,ix,iz-1)];
		WL[q]   = d_W4[Id(iy,ix,iz)];
		WR[q]   = d_W4[Id(iy,ix,iz+1)];
		WRR[q]  = d_W4[Id(iy,ix,iz+2)];
		WRRR[q] = d_W4[Id(iy,ix,iz+3)];
		
		WLLLN[q] = d_W4[Id(iy,ix+1,iz-2)];
		WLLN[q]  = d_W4[Id(iy,ix+1,iz-1)];
		WLN[q]   = d_W4[Id(iy,ix+1,iz)];
		WRN[q]   = d_W4[Id(iy,ix+1,iz+1)];
		WRRN[q]  = d_W4[Id(iy,ix+1,iz+2)];
		WRRRN[q] = d_W4[Id(iy,ix+1,iz+3)];
		
		WLLLS[q] = d_W4[Id(iy,ix-1,iz-2)];
		WLLS[q]  = d_W4[Id(iy,ix-1,iz-1)];
		WLS[q]   = d_W4[Id(iy,ix-1,iz)];
		WRS[q]   = d_W4[Id(iy,ix-1,iz+1)];
		WRRS[q]  = d_W4[Id(iy,ix-1,iz+2)];
		WRRRS[q] = d_W4[Id(iy,ix-1,iz+3)];

		WLLLF[q] = d_W4[Id(iy+1,ix,iz-2)];
		WLLF[q]  = d_W4[Id(iy+1,ix,iz-1)];
		WLF[q]   = d_W4[Id(iy+1,ix,iz)];
		WRF[q]   = d_W4[Id(iy+1,ix,iz+1)];
		WRRF[q]  = d_W4[Id(iy+1,ix,iz+2)];
		WRRRF[q] = d_W4[Id(iy+1,ix,iz+3)];

		WLLLB[q] = d_W4[Id(iy-1,ix,iz-2)];
		WLLB[q]  = d_W4[Id(iy-1,ix,iz-1)];
		WLB[q]   = d_W4[Id(iy-1,ix,iz)];
		WRB[q]   = d_W4[Id(iy-1,ix,iz+1)];
		WRRB[q]  = d_W4[Id(iy-1,ix,iz+2)];
		WRRRB[q] = d_W4[Id(iy-1,ix,iz+3)];
 
///////////////////////////////////////////////////////

		d_ApplyWENO(WLLL, WLL, WL, WR, WRR, WRRR, Wl, Wr);
		d_ApplyWENO(WLLLN, WLLN, WLN, WRN, WRRN, WRRRN, WlN, WrN);
		d_ApplyWENO(WLLLS, WLLS, WLS, WRS, WRRS, WRRRS, WlS, WrS);
		d_ApplyWENO(WLLLF, WLLF, WLF, WRF, WRRF, WRRRF, WlF, WrF);
		d_ApplyWENO(WLLLB, WLLB, WLB, WRB, WRRB, WRRRB, WlB, WrB);
		
		d_Wl0[Id(iy,ix,iz)] = Wl[0];
		d_Wl1[Id(iy,ix,iz)] = Wl[1];
		d_Wl2[Id(iy,ix,iz)] = Wl[2];
		d_Wl3[Id(iy,ix,iz)] = Wl[3];
		d_Wl4[Id(iy,ix,iz)] = Wl[4];
		
		d_Wr0[Id(iy,ix,iz)] = Wr[0];
		d_Wr1[Id(iy,ix,iz)] = Wr[1];
		d_Wr2[Id(iy,ix,iz)] = Wr[2];
		d_Wr3[Id(iy,ix,iz)] = Wr[3];
		d_Wr4[Id(iy,ix,iz)] = Wr[4];
		
		d_DW0xl[Id(iy,ix,iz)] = 2*(Wl[0] - WL[0]) / d_dx;
		d_DW1xl[Id(iy,ix,iz)] = 2*(Wl[1] - WL[1]) / d_dx;
		d_DW2xl[Id(iy,ix,iz)] = 2*(Wl[2] - WL[2]) / d_dx;
		d_DW3xl[Id(iy,ix,iz)] = 2*(Wl[3] - WL[3]) / d_dx;
		d_DW4xl[Id(iy,ix,iz)] = 2*(Wl[4] - WL[4]) / d_dx;
	
		d_DW0xr[Id(iy,ix,iz)] = 2*(WR[0] - Wr[0]) / d_dx;
		d_DW1xr[Id(iy,ix,iz)] = 2*(WR[1] - Wr[1]) / d_dx;
		d_DW2xr[Id(iy,ix,iz)] = 2*(WR[2] - Wr[2]) / d_dx;
		d_DW3xr[Id(iy,ix,iz)] = 2*(WR[3] - Wr[3]) / d_dx;
		d_DW4xr[Id(iy,ix,iz)] = 2*(WR[4] - Wr[4]) / d_dx;
		
		d_DW0yl[Id(iy,ix,iz)] = 0.5*(WlN[0] - WlS[0]) / d_dx;
		d_DW1yl[Id(iy,ix,iz)] = 0.5*(WlN[1] - WlS[1]) / d_dx;
		d_DW2yl[Id(iy,ix,iz)] = 0.5*(WlN[2] - WlS[2]) / d_dx;
		d_DW3yl[Id(iy,ix,iz)] = 0.5*(WlN[3] - WlS[3]) / d_dx;
		d_DW4yl[Id(iy,ix,iz)] = 0.5*(WlN[4] - WlS[4]) / d_dx;
		
		d_DW0yr[Id(iy,ix,iz)] = 0.5*(WrN[0] - WrS[0]) / d_dx;
		d_DW1yr[Id(iy,ix,iz)] = 0.5*(WrN[1] - WrS[1]) / d_dx;
		d_DW2yr[Id(iy,ix,iz)] = 0.5*(WrN[2] - WrS[2]) / d_dx;
		d_DW3yr[Id(iy,ix,iz)] = 0.5*(WrN[3] - WrS[3]) / d_dx;
		d_DW4yr[Id(iy,ix,iz)] = 0.5*(WrN[4] - WrS[4]) / d_dx;

		d_DW0zl[Id(iy,ix,iz)] = 0.5*(WlF[0] - WlB[0]) / d_dx;
		d_DW1zl[Id(iy,ix,iz)] = 0.5*(WlF[1] - WlB[1]) / d_dx;
		d_DW2zl[Id(iy,ix,iz)] = 0.5*(WlF[2] - WlB[2]) / d_dx;
		d_DW3zl[Id(iy,ix,iz)] = 0.5*(WlF[3] - WlB[3]) / d_dx;
		d_DW4zl[Id(iy,ix,iz)] = 0.5*(WlF[4] - WlB[4]) / d_dx;
		
		d_DW0zr[Id(iy,ix,iz)] = 0.5*(WrF[0] - WrB[0]) / d_dx;
		d_DW1zr[Id(iy,ix,iz)] = 0.5*(WrF[1] - WrB[1]) / d_dx;
		d_DW2zr[Id(iy,ix,iz)] = 0.5*(WrF[2] - WrB[2]) / d_dx;
		d_DW3zr[Id(iy,ix,iz)] = 0.5*(WrF[3] - WrB[3]) / d_dx;
		d_DW4zr[Id(iy,ix,iz)] = 0.5*(WrF[4] - WrB[4]) / d_dx;
	}
}

/**-------------------------------------------------------------------------------------------------*
 * Global Function : Flips the momentum and derivatives [for flux calculation in Y and Z direction]
 *--------------------------------------------------------------------------------------------------*/
__global__ void flip_Kernel(ptype *d_Wl1, ptype *d_Wl2, ptype *d_DW1xl, ptype *d_DW2xl, ptype *d_DW1yl, ptype *d_DW2yl, ptype *d_DW1zl, ptype *d_DW2zl,
                            ptype *d_Wr1, ptype *d_Wr2, ptype *d_DW1xr, ptype *d_DW2xr, ptype *d_DW1yr, ptype *d_DW2yr, ptype *d_DW1zr, ptype *d_DW2zr)

{
	int I = blockIdx.x*(blockDim.x) + threadIdx.x;

	int iy =  I/(d_nt_segx*d_nt_segz);
	int ix =  (I%(d_nt_segx*d_nt_segz))/d_nt_segz;
	int iz =  I%d_nt_segz;
	
	ptype temp;
			
	if(ix < d_nt_segx && iy < d_nt_segy && iz < d_nt_segz) {
		
		temp = d_Wl1[I];   d_Wl1[I] = d_Wl2[I];     d_Wl2[I] = temp;	
		temp = d_Wr1[I];   d_Wr1[I] = d_Wr2[I];     d_Wr2[I] = temp;	
		temp = d_DW1xl[I]; d_DW1xl[I] = d_DW2xl[I]; d_DW2xl[I] = temp;	
		temp = d_DW1yl[I]; d_DW1yl[I] = d_DW2yl[I]; d_DW2yl[I] = temp;	
		temp = d_DW1zl[I]; d_DW1zl[I] = d_DW2zl[I]; d_DW2zl[I] = temp;	
		temp = d_DW1xr[I]; d_DW1xr[I] = d_DW2xr[I]; d_DW2xr[I] = temp;	
		temp = d_DW1yr[I]; d_DW1yr[I] = d_DW2yr[I]; d_DW2yr[I] = temp;	
		temp = d_DW1zr[I]; d_DW1zr[I] = d_DW2zr[I]; d_DW2zr[I] = temp;	
	}	
}

/**-----------------------------------------------------------------------------------------*
 * Global Function : Flips back the Flux
 *------------------------------------------------------------------------------------------*/
__global__ void flipBack_Kernel(ptype *d_F1, ptype *d_F2)
{
	int I = blockIdx.x*(blockDim.x) + threadIdx.x;

	int iy =  I/(d_nt_segx*d_nt_segz);
	int ix =  (I%(d_nt_segx*d_nt_segz))/d_nt_segz;
	int iz =  I%d_nt_segz;
    
    ptype temp;
	
	if(ix < d_nt_segx && iy < d_nt_segy && iz < d_nt_segz) {
		temp = d_F1[I]; d_F1[I] = d_F2[I]; d_F2[I] = temp;
	}
}

/**-----------------------------------------------------------------------------------------*
 * Global Function : Updates the flow field in every iteration
 *------------------------------------------------------------------------------------------*/

__global__ void update_Kernel (ptype *d_W, ptype *d_Fx, ptype *d_Fy, ptype *d_Fz)
{
	int I = blockIdx.x*blockDim.x + threadIdx.x;

	int iy =  I/(d_nt_segx*d_nt_segz);
	int ix =  (I%(d_nt_segx*d_nt_segz))/d_nt_segz;
	int iz =  I%d_nt_segz;

	if(ix < d_nt_segx-3 && iy < d_nt_segy-3 && iz < d_nt_segz-3 && ix > 2 && iy > 2 && iz > 2)
	{
		
		d_W[I] = d_W[I] - (1/d_dx) * ( d_Fx[Id(iy, ix, iz)] - d_Fx[Id(iy, ix-1, iz)] +
                                       d_Fy[Id(iy, ix, iz)] - d_Fy[Id(iy-1, ix, iz)] +
									   d_Fz[Id(iy, ix, iz)] - d_Fz[Id(iy, ix, iz-1)] );		
	}
}

/**-----------------------------------------------------------------------------------------*
 * Global Function : Calculates Flux
 *------------------------------------------------------------------------------------------*/
__global__ void flux(ptype *d_W0, ptype *d_W1, ptype *d_W2, ptype *d_W3, ptype *d_W4,
					 ptype *d_Wl0, ptype *d_Wl1, ptype *d_Wl2, ptype *d_Wl3, ptype *d_Wl4,
					 ptype *d_Wr0, ptype *d_Wr1, ptype *d_Wr2, ptype *d_Wr3, ptype *d_Wr4,
					 ptype *d_DW0xl, ptype *d_DW1xl, ptype *d_DW2xl, ptype *d_DW3xl, ptype *d_DW4xl,
					 ptype *d_DW0xr, ptype *d_DW1xr, ptype *d_DW2xr, ptype *d_DW3xr, ptype *d_DW4xr,
					 ptype *d_DW0yl, ptype *d_DW1yl, ptype *d_DW2yl, ptype *d_DW3yl, ptype *d_DW4yl,
					 ptype *d_DW0yr, ptype *d_DW1yr, ptype *d_DW2yr, ptype *d_DW3yr, ptype *d_DW4yr,
					 ptype *d_DW0zl, ptype *d_DW1zl, ptype *d_DW2zl, ptype *d_DW3zl, ptype *d_DW4zl,
					 ptype *d_DW0zr, ptype *d_DW1zr, ptype *d_DW2zr, ptype *d_DW3zr, ptype *d_DW4zr,
					 ptype *d_F0, ptype *d_F1, ptype *d_F2, ptype *d_F3, ptype *d_F4, int TAG)

{
	int Id = blockIdx.x*blockDim.x + threadIdx.x;

	int iy =  Id/(d_nt_segx*d_nt_segz);
	int ix =  (Id%(d_nt_segx*d_nt_segz))/d_nt_segz;
	int iz =  Id%d_nt_segz;
	
/* ----------------------------------------------------------------------------------------------------------------------------------*
 * Variables Decleration
 * ----------------------------------------------------------------------------------------------------------------------------------*/	
	ptype Pl, denl, laml, Ul[3], Pr, denr, lamr, Ur[3], We[5], Pe, dene, lame, Ue[3], PL, TL,  denL, UL[3];	
	ptype Ie2l, Ie4l, Ie2r, Ie4r, Ipl[3][7], Ifl[3][7], Inr[3][7], Ifr[3][7], Ie2e, Ie4e, Ipe[3][7], Ine[3][7], Ife[3][7];

	ptype Mpgl000[5], Mngr000[5];
	ptype Mfgl100_axl[5], Mfgr100_axr[5], Mfgl010_ayl[5], Mfgr010_ayr[5], Mfgl001_azl[5], Mfgr001_azr[5];

	ptype bxl[5], bxr[5], byl[5], byr[5], bzl[5], bzr[5], bx_l[5], bx_r[5], by_[5], bz_[5], Bl[5], Br[5], B_[5];
	ptype axl[5], axr[5], ayl[5], ayr[5], azl[5], azr[5], ax_l[5], ax_r[5], ay_[5], az_[5], Al[5], Ar[5], A_[5];

	ptype Mpgl000_ayl[5], Mngr000_ayr[5], Mpgl000_azl[5], Mngr000_azr[5];
	ptype tau, w, gm0, gm1, gm2, gm3, gm4, gm5;

	ptype Mfge000[5], Mpge100_ax_l[5], Mnge100_ax_r[5], Mfge010_ay_[5], Mfge001_az_[5], Mpgl100_axl[5], 
          Mngr100_axr[5], Mpgl010_ayl[5], Mngr010_ayr[5], Mpgl001_azl[5], Mngr001_azr[5], Mpgl000_Al[5], Mngr000_Ar[5];

	ptype P, Q, p0, p1, p2, p3, p4, p5;

	ptype Mfge100[5], Mpgl100[5], Mngr100[5], Mpge200_ax_l[5], Mfge100_A_[5], Mnge200_ax_r[5], Mfge110_ay_[5], Mfge101_az_[5], 
	      Mpgl200_axl[5], Mngr200_axr[5], Mpgl110_ayl[5], Mngr110_ayr[5], Mpgl101_azl[5], Mngr101_azr[5], Mpgl100_Al[5], Mngr100_Ar[5];		
	
	ptype WL[5], WR[5];
	ptype Wl[5], Wr[5];
		
	
	bool bound_check;
	if      (TAG == 1) {bound_check = (ix < d_nt_segx-3 && iy < d_nt_segy-3 && iz < d_nt_segz-3 && ix > 1 && iy > 2 && iz > 2);}
	else if (TAG == 2) {bound_check = (ix < d_nt_segx-3 && iy < d_nt_segy-3 && iz < d_nt_segz-3 && ix > 2 && iy > 1 && iz > 2);}
	else if (TAG == 3) {bound_check = (ix < d_nt_segx-3 && iy < d_nt_segy-3 && iz < d_nt_segz-3 && ix > 2 && iy > 2 && iz > 1);}
	
	if (bound_check) {
		// X-Flux
		if (TAG == 1) {
			WL[0] = d_W0[Id(iy,ix,iz)];
			WL[1] = d_W1[Id(iy,ix,iz)];
			WL[2] = d_W2[Id(iy,ix,iz)];
			WL[3] = d_W3[Id(iy,ix,iz)];
			WL[4] = d_W4[Id(iy,ix,iz)];

			WR[0] = d_W0[Id(iy,ix+1,iz)];
			WR[1] = d_W1[Id(iy,ix+1,iz)];
			WR[2] = d_W2[Id(iy,ix+1,iz)];
			WR[3] = d_W3[Id(iy,ix+1,iz)];
			WR[4] = d_W4[Id(iy,ix+1,iz)];

			Wl[0] = d_Wl0[Id(iy,ix,iz)];
			Wl[1] = d_Wl1[Id(iy,ix,iz)];
			Wl[2] = d_Wl2[Id(iy,ix,iz)];
			Wl[3] = d_Wl3[Id(iy,ix,iz)];
			Wl[4] = d_Wl4[Id(iy,ix,iz)];

			Wr[0] = d_Wr0[Id(iy,ix,iz)];
			Wr[1] = d_Wr1[Id(iy,ix,iz)];
			Wr[2] = d_Wr2[Id(iy,ix,iz)];
			Wr[3] = d_Wr3[Id(iy,ix,iz)];
			Wr[4] = d_Wr4[Id(iy,ix,iz)];		

		}
		// Y-Flux
		if (TAG == 2) {
			WL[0] = d_W0[Id(iy,ix,iz)];
			WL[2] = d_W1[Id(iy,ix,iz)];
			WL[1] = d_W2[Id(iy,ix,iz)];
			WL[3] = d_W3[Id(iy,ix,iz)];
			WL[4] = d_W4[Id(iy,ix,iz)];

			WR[0] = d_W0[Id(iy+1,ix,iz)];
			WR[2] = d_W1[Id(iy+1,ix,iz)];
			WR[1] = d_W2[Id(iy+1,ix,iz)];
			WR[3] = d_W3[Id(iy+1,ix,iz)];
			WR[4] = d_W4[Id(iy+1,ix,iz)];

			Wl[0] = d_Wl0[Id(iy,ix,iz)];
			Wl[1] = d_Wl1[Id(iy,ix,iz)];
			Wl[2] = d_Wl2[Id(iy,ix,iz)];
			Wl[3] = d_Wl3[Id(iy,ix,iz)];
			Wl[4] = d_Wl4[Id(iy,ix,iz)];

			Wr[0] = d_Wr0[Id(iy,ix,iz)];
			Wr[1] = d_Wr1[Id(iy,ix,iz)];
			Wr[2] = d_Wr2[Id(iy,ix,iz)];
			Wr[3] = d_Wr3[Id(iy,ix,iz)];
			Wr[4] = d_Wr4[Id(iy,ix,iz)];
		}
		// Z-Flux
		if (TAG == 3) {
			WL[0] = d_W0[Id(iy,ix,iz)];
			WL[3] = d_W1[Id(iy,ix,iz)];
			WL[2] = d_W2[Id(iy,ix,iz)];
			WL[1] = d_W3[Id(iy,ix,iz)];
			WL[4] = d_W4[Id(iy,ix,iz)];

			WR[0] = d_W0[Id(iy,ix,iz+1)];
			WR[3] = d_W1[Id(iy,ix,iz+1)];
			WR[2] = d_W2[Id(iy,ix,iz+1)];
			WR[1] = d_W3[Id(iy,ix,iz+1)];
			WR[4] = d_W4[Id(iy,ix,iz+1)];			

			Wl[0] = d_Wl0[Id(iy,ix,iz)];
			Wl[1] = d_Wl1[Id(iy,ix,iz)];
			Wl[2] = d_Wl2[Id(iy,ix,iz)];
			Wl[3] = d_Wl3[Id(iy,ix,iz)];
			Wl[4] = d_Wl4[Id(iy,ix,iz)];

			Wr[0] = d_Wr0[Id(iy,ix,iz)];
			Wr[1] = d_Wr1[Id(iy,ix,iz)];
			Wr[2] = d_Wr2[Id(iy,ix,iz)];
			Wr[3] = d_Wr3[Id(iy,ix,iz)];
			Wr[4] = d_Wr4[Id(iy,ix,iz)];
		}
		
/* ------------------------------------------------------------------*
 * INTEGRATIONS (MOMENT CALCULATIONS) [l & r]
 * ------------------------------------------------------------------*/
		d_c2p(Wl, denl, Ul, Pl);	d_c2p(Wr, denr, Ur, Pr);
		laml  = 0.5*denl/Pl;	lamr  = 0.5*denr/Pr;
	
		Ie2l = d_K/(2*laml);
		Ie4l = 3*d_K/(4*laml*laml) + d_K*(d_K-1)/(4*laml*laml);
		Ie2r = d_K/(2*lamr);
		Ie4r = 3*d_K/(4*lamr*lamr) + d_K*(d_K-1)/(4*lamr*lamr);
	
		for(int j=0; j<3; j++)
		{
			Ifl[j][0] = 1.0;
			Ifl[j][1] = Ul[j];
			Ifr[j][0] = 1.0;
			Ifr[j][1] = Ur[j];
			Ipl[j][0] = 0.5*(erfc(-sqrt(laml)*Ul[j]));
			Inr[j][0] = 0.5*(erfc(sqrt(lamr)*Ur[j]));
			Ipl[j][1] = Ul[j]*Ipl[j][0] + 0.5*exp(-laml*Ul[j]*Ul[j])/sqrt(laml*pi);
			Inr[j][1] = Ur[j]*Inr[j][0] - 0.5*exp(-lamr*Ur[j]*Ur[j])/sqrt(lamr*pi);

			for(int i=2; i<7; i++)
			{
				Ipl[j][i] = Ul[j]*Ipl[j][i-1] + Ipl[j][i-2]*(i-1)/(2*laml);
				Ifl[j][i] = Ul[j]*Ifl[j][i-1] + Ifl[j][i-2]*(i-1)/(2*laml);
				Inr[j][i] = Ur[j]*Inr[j][i-1] + Inr[j][i-2]*(i-1)/(2*lamr);
				Ifr[j][i] = Ur[j]*Ifr[j][i-1] + Ifr[j][i-2]*(i-1)/(2*lamr);
			}
		}

/* ------------------------------------------------------------------*
 * W0 CALCULATION
 * ------------------------------------------------------------------*/
		d_MCal(Mpgl000, Ipl, Ifl, Ie2l, Ie4l, 0, 0, 0);
		d_MCal(Mngr000, Inr, Ifr, Ie2r, Ie4r, 0, 0, 0);
	
		FOR(q, 5)
			We[q] = denl * Mpgl000[q] + denr * Mngr000[q];
	
		d_c2p(We, dene, Ue, Pe);	lame  = 0.5*dene/Pe;

/* ------------------------------------------------------------------*
 * INTEGRATIONS (MOMENT CALCULATIONS) [e]
 * ------------------------------------------------------------------*/
		Ie2e = d_K/(2*lame);
		Ie4e = 3*d_K/(4*lame*lame) + d_K*(d_K-1)/(4*lame*lame);
	
		for(int j=0; j<3; j++) {
			Ife[j][0] = 1.0;
			Ife[j][1] = Ue[j];
			Ipe[j][0] = 0.5*(erfc(-sqrt(lame)*Ue[j]));
			Ine[j][0] = 0.5*(erfc(sqrt(lame)*Ue[j]));
			Ipe[j][1] = Ue[j]*Ipe[j][0] + 0.5*exp(-lame*Ue[j]*Ue[j])/sqrt(lame*pi);
			Ine[j][1] = Ue[j]*Ine[j][0] - 0.5*exp(-lame*Ue[j]*Ue[j])/sqrt(lame*pi);

			for(int i=2; i<7; i++) {
				Ipe[j][i] = Ue[j]*Ipe[j][i-1] + Ipe[j][i-2]*(i-1)/(2*lame);
				Ife[j][i] = Ue[j]*Ife[j][i-1] + Ife[j][i-2]*(i-1)/(2*lame);
				Ine[j][i] = Ue[j]*Ine[j][i-1] + Ine[j][i-2]*(i-1)/(2*lame);
			}	
		}
/* --------------------------------------------------------------------------*
 * Slope Calculation [phase I]: (axl, axr, ayl, ayr, azl, azr, ax_l, ax_r)
 * --------------------------------------------------------------------------*/
		int x = 0;
			bxl[x] = d_DW0xl[Id]/denl;
			bxr[x] = d_DW0xr[Id]/denr;		
			byl[x] = d_DW0yl[Id]/denl;
			byr[x] = d_DW0yr[Id]/denr;		
			bzl[x] = d_DW0zl[Id]/denl;		
			bzr[x] = d_DW0zr[Id]/denr;
			bx_l[x] = 2*(We[x] - WL[x]) / (dene*d_dx);
			bx_r[x] = 2*(WR[x] - We[x]) / (dene*d_dx);
			
			x = 1;
			bxl[x] = d_DW1xl[Id]/denl;
			bxr[x] = d_DW1xr[Id]/denr;		
			byl[x] = d_DW1yl[Id]/denl;
			byr[x] = d_DW1yr[Id]/denr;		
			bzl[x] = d_DW1zl[Id]/denl;		
			bzr[x] = d_DW1zr[Id]/denr;		
			bx_l[x] = 2*(We[x] - WL[x]) / (dene*d_dx);
			bx_r[x] = 2*(WR[x] - We[x]) / (dene*d_dx);
			
			x = 2;
			bxl[x] = d_DW2xl[Id]/denl;
			bxr[x] = d_DW2xr[Id]/denr;		
			byl[x] = d_DW2yl[Id]/denl;
			byr[x] = d_DW2yr[Id]/denr;		
			bzl[x] = d_DW2zl[Id]/denl;		
			bzr[x] = d_DW2zr[Id]/denr;		
			bx_l[x] = 2*(We[x] - WL[x]) / (dene*d_dx);
			bx_r[x] = 2*(WR[x] - We[x]) / (dene*d_dx);
			
			x = 3;
			bxl[x] = d_DW3xl[Id]/denl;
			bxr[x] = d_DW3xr[Id]/denr;		
			byl[x] = d_DW3yl[Id]/denl;
			byr[x] = d_DW3yr[Id]/denr;		
			bzl[x] = d_DW3zl[Id]/denl;		
			bzr[x] = d_DW3zr[Id]/denr;		
			bx_l[x] = 2*(We[x] - WL[x]) / (dene*d_dx);
			bx_r[x] = 2*(WR[x] - We[x]) / (dene*d_dx);
		
			x = 4;
			bxl[x] = d_DW4xl[Id]/denl;
			bxr[x] = d_DW4xr[Id]/denr;		
			byl[x] = d_DW4yl[Id]/denl;
			byr[x] = d_DW4yr[Id]/denr;		
			bzl[x] = d_DW4zl[Id]/denl;		
			bzr[x] = d_DW4zr[Id]/denr;		
			bx_l[x] = 2*(We[x] - WL[x]) / (dene*d_dx);
			bx_r[x] = 2*(WR[x] - We[x]) / (dene*d_dx);

		d_slopesolver(bxl, Ul, laml, axl);
		d_slopesolver(bxr, Ur, lamr, axr);
		d_slopesolver(byl, Ul, laml, ayl);
		d_slopesolver(byr, Ur, lamr, ayr);
		d_slopesolver(bzl, Ul, laml, azl);
		d_slopesolver(bzr, Ur, lamr, azr);
		d_slopesolver(bx_l, Ue, lame, ax_l);
		d_slopesolver(bx_r, Ue, lame, ax_r);

/* -------------------------------------------------------------------------*
 * Slope Calculation [phase II]: (ay_, az_, Al, Ar)
 * -------------------------------------------------------------------------*/
		d_MCal(Mpgl000_ayl, Ipl, Ifl, Ie2l, Ie4l, 0, 0, 0, ayl);
		d_MCal(Mngr000_ayr, Inr, Ifr, Ie2r, Ie4r, 0, 0, 0, ayr);
		d_MCal(Mpgl000_azl, Ipl, Ifl, Ie2l, Ie4l, 0, 0, 0, azl);
		d_MCal(Mngr000_azr, Inr, Ifr, Ie2r, Ie4r, 0, 0, 0, azr);
		d_MCal(Mfgl100_axl, Ifl, Ifl, Ie2l, Ie4l, 1, 0, 0, axl);
		d_MCal(Mfgr100_axr, Ifr, Ifr, Ie2r, Ie4r, 1, 0, 0, axr);
		d_MCal(Mfgl010_ayl, Ifl, Ifl, Ie2l, Ie4l, 0, 1, 0, ayl);
		d_MCal(Mfgr010_ayr, Ifr, Ifr, Ie2r, Ie4r, 0, 1, 0, ayr);
		d_MCal(Mfgl001_azl, Ifl, Ifl, Ie2l, Ie4l, 0, 0, 1, azl);
		d_MCal(Mfgr001_azr, Ifr, Ifr, Ie2r, Ie4r, 0, 0, 1, azr);
	
		FOR(i, 5) {
			by_[i] = Mpgl000_ayl[i] + Mngr000_ayr[i];
			bz_[i] = Mpgl000_azl[i] + Mngr000_azr[i];	
			Bl[i] = -Mfgl100_axl[i] - Mfgl010_ayl[i] - Mfgl001_azl[i];		 
			Br[i] = -Mfgr100_axr[i] - Mfgr010_ayr[i] - Mfgr001_azr[i];		 
		}
		d_slopesolver(by_, Ue, lame, ay_);
		d_slopesolver(bz_, Ue, lame, az_);
		d_slopesolver(Bl, Ul, laml, Al);
		d_slopesolver(Br, Ur, lamr, Ar);
	
/* -------------------------------------------------------------------------*
 * Collision time scale calculation (tau)
 * -------------------------------------------------------------------------*/
		d_c2p(WL, denL, UL, PL);
		TL   = PL/(R*denL);
		
		tau = d_mu0 * ( pow((TL/d_T0),1.5)*(d_T0 + 110.4)/(TL + 110.4) ) / PL;

		w = abs(denl/laml - denr/lamr)/(abs(denl/laml + denr/lamr));
		tau = tau + d_dt*w;
	
/* -------------------------------------------------------------------------*
 * Slope Calculation [phase III]: (A_)
 * -------------------------------------------------------------------------*/
		gm0 = d_dt - tau*(1-exp(-d_dt/tau));	gm1 = -(1-exp(-d_dt/tau))/gm0;
		gm2 = (-d_dt+2*tau*(1-exp(-d_dt/tau))-d_dt*exp(-d_dt/tau))/gm0;	gm3 = -gm1;
		gm4 = (d_dt*exp(-d_dt/tau) -tau*(1-exp(-d_dt/tau)))/gm0;		gm5 = tau*gm3;
		
		d_MCal(Mfge000, Ife, Ife, Ie2e, Ie4e, 0, 0, 0);
		d_MCal(Mpge100_ax_l, Ipe, Ife, Ie2e, Ie4e, 1, 0, 0, ax_l);	
		d_MCal(Mnge100_ax_r, Ine, Ife, Ie2e, Ie4e, 1, 0, 0, ax_r);	
		d_MCal(Mfge010_ay_,  Ife, Ife, Ie2e, Ie4e, 0, 1, 0, ay_);	
		d_MCal(Mfge001_az_,  Ife, Ife, Ie2e, Ie4e, 0, 0, 1, az_);	
		d_MCal(Mpgl100_axl,  Ipl, Ifl, Ie2l, Ie4l, 1, 0, 0, axl);	
		d_MCal(Mngr100_axr,  Inr, Ifr, Ie2r, Ie4r, 1, 0, 0, axr);	
		d_MCal(Mpgl010_ayl,  Ipl, Ifl, Ie2l, Ie4l, 0, 1, 0, ayl);	
		d_MCal(Mngr010_ayr,  Inr, Ifr, Ie2r, Ie4r, 0, 1, 0, ayr);	
		d_MCal(Mpgl001_azl,  Ipl, Ifl, Ie2l, Ie4l, 0, 0, 1, azl);	
		d_MCal(Mngr001_azr,  Inr, Ifr, Ie2r, Ie4r, 0, 0, 1, azr);	
		d_MCal(Mpgl000_Al,   Ipl, Ifl, Ie2l, Ie4l, 0, 0, 0, Al);	
		d_MCal(Mngr000_Ar,   Inr, Ifr, Ie2r, Ie4r, 0, 0, 0, Ar);
		
		FOR(i, 5) {
			B_[i] = gm1*dene*Mfge000[i] + gm2*dene*(Mpge100_ax_l[i]+Mnge100_ax_r[i]+Mfge010_ay_[i]+Mfge001_az_[i]) + gm3*(denl*Mpgl000[i]+denr*Mngr000[i]) +
					(gm4+gm5)*( denl*(Mpgl100_axl[i]+Mpgl010_ayl[i]+Mpgl001_azl[i]) + denr*(Mngr100_axr[i]+Mngr010_ayr[i]+Mngr001_azr[i]) ) +
					gm5*(denl*Mpgl000_Al[i]+denr*Mngr000_Ar[i]);
		       		
			B_[i] = B_[i] / dene;
		}
		d_slopesolver(B_, Ue, lame, A_);
	
/* ---------------------------------------------------------------------------------------------------------*
 * Flux calculation	
 * ---------------------------------------------------------------------------------------------------------*/
//		Integral dt
		P  = -tau*(exp(-d_dt/tau)-1.0);
		Q  = -tau*d_dt*exp(-d_dt/tau)-tau*tau*(exp(-d_dt/tau)-1.0);
		p0 = (d_dt-P);			p1 = 0.5*d_dt*d_dt - tau*(p0);
		p2 = -tau*(p0) + Q;		p3 = P;
		p4 = -Q - tau*P;		p5 = -tau*P;
	
		d_MCal(Mpgl100, Ipl, Ifl, Ie2l, Ie4l, 1, 0, 0);
		d_MCal(Mngr100, Inr, Ifr, Ie2r, Ie4r, 1, 0, 0);
		d_MCal(Mfge100, Ife, Ife, Ie2e, Ie4e, 1, 0, 0);
		d_MCal(Mfge100_A_,   Ife, Ife, Ie2e, Ie4e, 1, 0, 0, A_);	
		d_MCal(Mpge200_ax_l, Ipe, Ife, Ie2e, Ie4e, 2, 0, 0, ax_l);	
		d_MCal(Mnge200_ax_r, Ine, Ife, Ie2e, Ie4e, 2, 0, 0, ax_r);	
		d_MCal(Mfge110_ay_,  Ife, Ife, Ie2e, Ie4e, 1, 1, 0, ay_);	
		d_MCal(Mfge101_az_,  Ife, Ife, Ie2e, Ie4e, 1, 0, 1, az_);	
		d_MCal(Mpgl200_axl,  Ipl, Ifl, Ie2l, Ie4l, 2, 0, 0, axl);	
		d_MCal(Mngr200_axr,  Inr, Ifr, Ie2r, Ie4r, 2, 0, 0, axr);	
		d_MCal(Mpgl110_ayl,  Ipl, Ifl, Ie2l, Ie4l, 1, 1, 0, ayl);	
		d_MCal(Mngr110_ayr,  Inr, Ifr, Ie2r, Ie4r, 1, 1, 0, ayr);	
		d_MCal(Mpgl101_azl,  Ipl, Ifl, Ie2l, Ie4l, 1, 0, 1, azl);	
		d_MCal(Mngr101_azr,  Inr, Ifr, Ie2r, Ie4r, 1, 0, 1, azr);	
		d_MCal(Mpgl100_Al,   Ipl, Ifl, Ie2l, Ie4l, 1, 0, 0, Al);	
		d_MCal(Mngr100_Ar,   Inr, Ifr, Ie2r, Ie4r, 1, 0, 0, Ar);	
		ptype F[5] = {0};
		
		FOR(i, 5) {
			F[i] = p0*dene*Mfge100[i] + p1*dene*Mfge100_A_[i] + p2*dene*(Mpge200_ax_l[i]+Mnge200_ax_r[i]+Mfge110_ay_[i]+Mfge101_az_[i]) + 
				   p3*(denl*Mpgl100[i]+denr*Mngr100[i]) +  p4*( denl*(Mpgl200_axl[i]+Mpgl110_ayl[i]+Mpgl101_azl[i]) + denr*(Mngr200_axr[i]+Mngr110_ayr[i]+Mngr101_azr[i]) ) + 
				   p5*(denl*Mpgl100_Al[i]+denr*Mngr100_Ar[i]);		
		}
	
		d_F0[Id] = F[0];
		d_F1[Id] = F[1];
		d_F2[Id] = F[2];
		d_F3[Id] = F[3];
		d_F4[Id] = F[4]; 		
		
	} // end of if statement for checking thread out of range access

}

	
/* ----------------------------------------------------------------------*
 *  SLOPE SOLVER
 * ----------------------------------------------------------------------*/

__device__ void d_slopesolver(ptype b[5], ptype U[3], ptype lam, ptype a[5])
{
	ptype R2, R3, R4, R5;
	
	R2 = b[1] - U[0]*b[0];
	R3 = b[2] - U[1]*b[0];
	R4 = b[3] - U[2]*b[0];
	R5 = 2*b[4] - b[0]*(U[0]*U[0]+U[1]*U[1]+U[2]*U[2]+(d_K+3)/(2*lam));
	
	a[4] = (1/PRN)*(R5-2*U[0]*R2-2*U[1]*R3-2*U[2]*R4)*(4*lam*lam)/(d_K+3);
	a[3] = 2*lam*R4 - U[2]*a[4];
	a[2] = 2*lam*R3 - U[1]*a[4];
	a[1] = 2*lam*R2 - U[0]*a[4];
	a[0] = b[0] - a[1]*U[0] - a[2]*U[1] -a[3]*U[2]-.5*a[4]*(U[0]*U[0] +
	       U[1]*U[1] + U[2]*U[2]+(d_K+3)/(2*lam));
}

/**-------------------------------------------------------------------------------------------------*
 * Device Function : Moment Integral Calculator
 *--------------------------------------------------------------------------------------------------*/
/* ---------------------------------------------------------------------------------------------------------------------------------------------------*
 *  Moment Matrix Calculator 
 * ---------------------------------------------------------------------------------------------------------------------------------------------------*/
__device__ void d_MCal(ptype M[5], ptype I[3][7], ptype If[3][7], ptype Ie2, ptype Ie4, int k, int l, int m, ptype ax[5])
{
	ptype val0, val1, val2, val3;
	 
	val0 = 0.5 * ( I[0][2+k]*If[1][l]*If[2][m] + I[0][k]*If[1][2+l]*If[2][m] + I[0][k]*If[1][l]*If[2][2+m] + I[0][k]*If[1][l]*If[2][m]*Ie2  );
	
	M[0] = ax[0]*(I[0][k]*If[1][l]*If[2][m]) + ax[1]*(I[0][1+k]*If[1][l]*If[2][m]) + ax[2]*(I[0][k]*If[1][1+l]*If[2][m]) +
	       ax[3]*(I[0][k]*If[1][l]*If[2][1+m]) + ax[4]*val0 ;	

				  
	val1 = 0.5 * ( I[0][3+k]*If[1][l]*If[2][m] + I[0][1+k]*If[1][2+l]*If[2][m] + I[0][1+k]*If[1][l]*If[2][2+m] +
	 			   I[0][1+k]*If[1][l]*If[2][m]*Ie2  );
	
	
	M[1] = ax[0]*(I[0][1+k]*If[1][l]*If[2][m]) + ax[1]*(I[0][2+k]*If[1][l]*If[2][m]) + ax[2]*(I[0][1+k]*If[1][1+l]*If[2][m]) +
		   ax[3]*(I[0][1+k]*If[1][l]*If[2][1+m]) + ax[4]*val1;
				 
 
   val2 = 0.5 * ( I[0][2+k]*If[1][1+l]*If[2][m] + I[0][k]*If[1][3+l]*If[2][m] + I[0][k]*If[1][1+l]*If[2][2+m] +
				  I[0][k]*If[1][1+l]*If[2][m]*Ie2  );
	
	
	M[2] = ax[0]*(I[0][k]*If[1][1+l]*If[2][m])   + ax[1]*(I[0][1+k]*If[1][1+l]*If[2][m]) + ax[2]*(I[0][k]*If[1][2+l]*If[2][m]) +
		   ax[3]*(I[0][k]*If[1][1+l]*If[2][1+m]) + ax[4]*val2;
				 
	
	val3 = 0.5 * ( I[0][2+k]*If[1][l]*If[2][1+m] + I[0][k]*If[1][2+l]*If[2][1+m] + I[0][k]*If[1][l]*If[2][3+m] + 
	               I[0][k]*If[1][l]*If[2][1+m]*Ie2  );
	
	
	M[3] = ax[0]*(I[0][k]*If[1][l]*If[2][1+m]) + ax[1]*(I[0][1+k]*If[1][l]*If[2][1+m]) + ax[2]*(I[0][k]*If[1][1+l]*If[2][1+m]) +
	       ax[3]*(I[0][k]*If[1][l]*If[2][2+m]) + ax[4]*val3;
				 
				 
	M[4] = 0.25*ax[4]* ( I[0][4+k]*If[1][l]*If[2][m] + I[0][k]*If[1][4+l]*If[2][m] + I[0][k]*If[1][l]*If[2][4+m] +
	                     I[0][k]*If[1][l]*If[2][m]*Ie4 + 2*I[0][2+k]*If[1][2+l]*If[2][m] + 2*I[0][2+k]*If[1][l]*If[2][2+m] +
	                   2*I[0][2+k]*If[1][l]*If[2][m]*Ie2 + 2*I[0][k]*If[1][2+l]*If[2][2+m] + 2*I[0][k]*If[1][2+l]*If[2][m]*Ie2 +
	                   2*I[0][k]*If[1][l]*If[2][2+m]*Ie2 ) +
		   ax[0]*val0 + ax[1]*val1 + ax[2]*val2 + ax[3]*val3;
}

__device__ void d_MCal(ptype M[5], ptype I[3][7], ptype If[3][7], ptype Ie2, ptype Ie4, int k, int l, int m)
{
    ptype val0	= 0.5 * ( I[0][2+k]*If[1][l]*If[2][m] + I[0][k]*If[1][2+l]*If[2][m] + I[0][k]*If[1][l]*If[2][2+m] + I[0][k]*If[1][l]*If[2][m]*Ie2  );
		
	M[0] = I[0][k]*If[1][l]*If[2][m];
	M[1] = I[0][1+k]*If[1][l]*If[2][m];
	M[2] = I[0][k]*If[1][1+l]*If[2][m];
	M[3] = I[0][k]*If[1][l]*If[2][1+m];
	M[4] = val0;
}

__device__ void d_c2p(ptype W[5], ptype &den, ptype U[3], ptype &P)
{
	den = W[0];
	U[0]  = W[1]/den;
	U[1]  = W[2]/den;
	U[2]  = W[3]/den;
	P   = (den*(GAM-1))*(W[4]/den - 0.5*(U[0]*U[0]+U[1]*U[1]+U[2]*U[2]));
}

/* ---------------------------------------------------------------------------------------------------------------------------------------------------*
 *  WENO - SYMOO
 * ---------------------------------------------------------------------------------------------------------------------------------------------------*/
__device__ void d_ApplyWENO(ptype WLLL[5], ptype WLL[5], ptype WL[5], ptype WR[5], ptype WRR[5],  ptype WRRR[5], ptype Wl[5], ptype Wr[5])
{	
	ptype P0p, P0n, P1p, P1n, P2p, P2n, P3p, P3n, IS0p, IS0n, IS1p, IS1n, IS2p, IS2n, IS3p, IS3n, 
	      gam0, gam1, gam2, gam3, ALPHA0p, ALPHA0n, ALPHA1p, ALPHA1n, ALPHA2p, ALPHA2n, ALPHA3p, ALPHA3n, sum_ALPHAp, sum_ALPHAn,
	      OMEGA0p, OMEGA0n, OMEGA1p, OMEGA1n, OMEGA2p, OMEGA2n, OMEGA3p, OMEGA3n, ISPMAX, ISNMAX;
	ptype TLLL, TLL, TL, TR, TRR, TRRR;
	gam0 = 1.0/20.0; gam1 = 9.0/20.0; gam2 = 9.0/20.0; gam3 = 1.0/20.0;
	
	FOR(i, 4) {
		P0p =     WLLL[i]/3  - 7*WLL[i]/6  + 11*WL[i]/6;
		P0n =     WRRR[i]/3  - 7*WRR[i]/6  + 11*WR[i]/6;
		P1p =    -WLL[i]/6   + 5*WL[i]/6   + WR[i]/3;
		P1n =    -WRR[i]/6   + 5*WR[i]/6   + WL[i]/3;	
		P2p =     WL[i]/3    + 5*WR[i]/6   - WRR[i]/6;
		P2n =     WR[i]/3    + 5*WL[i]/6   - WLL[i]/6;
		P3p =  11*WR[i]/6    - 7*WRR[i]/6  + WRRR[i]/3;
		P3n =  11*WL[i]/6    - 7*WLL[i]/6  + WLLL[i]/3;
			
		IS0p = 13*pow((WLLL[i] - 2*WLL[i] + WL[i]),2)/12    +  pow(( WLLL[i] - 4*WLL[i] + 3*WL[i]),2)/4;
		IS0n = 13*pow((WRRR[i] - 2*WRR[i] + WR[i]),2)/12    +  pow(( WRRR[i] - 4*WRR[i] + 3*WR[i]),2)/4;
		IS1p = 13*pow((WLL[i]  - 2*WL[i]  + WR[i]),2)/12    +  pow(( -WLL[i] + WR[i]),2)/4;
		IS1n = 13*pow((WRR[i]  - 2*WR[i]  + WL[i]),2)/12    +  pow(( -WRR[i] + WL[i]),2)/4;
		IS2p = 13*pow((WL[i]   - 2*WR[i]  + WRR[i]),2)/12   +  pow((-3*WL[i] + 4*WR[i] - WRR[i]),2)/4;
		IS2n = 13*pow((WR[i]   - 2*WL[i]  + WLL[i]),2)/12   +  pow((-3*WR[i] + 4*WL[i] - WLL[i]),2)/4;
		IS3p = 13*pow((WR[i]   - 2*WRR[i] + WRRR[i]),2)/12  +  pow((-5*WR[i] + 8*WRR[i] - 3*WRRR[i]),2)/4;
		IS3n = 13*pow((WL[i]   - 2*WLL[i] + WLLL[i]),2)/12  +  pow((-5*WL[i] + 8*WLL[i] - 3*WLLL[i]),2)/4;
		
		ISPMAX = max(IS0p,   IS1p);
		ISPMAX = max(ISPMAX, IS2p);
		ISPMAX = max(ISPMAX, IS3p);
		ISNMAX = max(IS0n,   IS1n);
		ISNMAX = max(ISNMAX, IS2n);
		ISNMAX = max(ISNMAX, IS3n);
		IS3p = ISPMAX; 
		IS3n = ISNMAX; 
				
		ALPHA0p = gam0*(1/(eps+IS0p)); 
		ALPHA0n = gam0*(1/(eps+IS0n)); 
		ALPHA1p = gam1*(1/(eps+IS1p)); 
		ALPHA1n = gam1*(1/(eps+IS1n)); 
		ALPHA2p = gam2*(1/(eps+IS2p)); 
		ALPHA2n = gam2*(1/(eps+IS2n)); 
		ALPHA3p = gam3*(1/(eps+IS3p)); 
		ALPHA3n = gam3*(1/(eps+IS3n)); 
	
		sum_ALPHAp = ALPHA0p + ALPHA1p + ALPHA2p + ALPHA3p;
		sum_ALPHAn = ALPHA0n + ALPHA1n + ALPHA2n + ALPHA3n;
	
		OMEGA0p = ALPHA0p/(sum_ALPHAp);
		OMEGA0n = ALPHA0n/(sum_ALPHAn);
		OMEGA1p = ALPHA1p/(sum_ALPHAp);
		OMEGA1n = ALPHA1n/(sum_ALPHAn);
		OMEGA2p = ALPHA2p/(sum_ALPHAp);
		OMEGA2n = ALPHA2n/(sum_ALPHAn);
		OMEGA3p = ALPHA3p/(sum_ALPHAp);
		OMEGA3n = ALPHA3n/(sum_ALPHAn);
	 
		Wl[i] = (OMEGA0p*P0p + OMEGA1p*P1p + OMEGA2p*P2p + OMEGA3p*P3p); 
		Wr[i] = (OMEGA0n*P0n + OMEGA1n*P1n + OMEGA2n*P2n + OMEGA3n*P3n); 
	}

		TL   = (GAM-1) * ( WL[4]/WL[0]     - 0.5*(WL[1]*WL[1] + WL[2]*WL[2] + WL[3]*WL[3])/(WL[0]*WL[0]) ) / R;
		TLL  = (GAM-1) * ( WLL[4]/WLL[0]   - 0.5*(WLL[1]*WLL[1] + WLL[2]*WLL[2] + WLL[3]*WLL[3])/(WLL[0]*WLL[0]) ) / R;
		TLLL = (GAM-1) * ( WLLL[4]/WLLL[0] - 0.5*(WLLL[1]*WLLL[1] + WLLL[2]*WLLL[2] + WLLL[3]*WLLL[3])/(WLLL[0]*WLLL[0]) ) / R;
		TR   = (GAM-1) * ( WR[4]/WR[0]     - 0.5*(WR[1]*WR[1] + WR[2]*WR[2] + WR[3]*WR[3])/(WR[0]*WR[0]) ) / R;
		TRR  = (GAM-1) * ( WRR[4]/WRR[0]   - 0.5*(WRR[1]*WRR[1] + WRR[2]*WRR[2] + WRR[3]*WRR[3])/(WRR[0]*WRR[0]) ) / R;
		TRRR = (GAM-1) * ( WRRR[4]/WRRR[0] - 0.5*(WRRR[1]*WRRR[1] + WRRR[2]*WRRR[2] + WRRR[3]*WRRR[3])/(WRRR[0]*WRRR[0]) ) / R;
		
		P0p =     TLLL/3  - 7*TLL/6  + 11*TL/6;
		P0n =     TRRR/3  - 7*TRR/6  + 11*TR/6;
		P1p =    -TLL/6   + 5*TL/6   + TR/3;
		P1n =    -TRR/6   + 5*TR/6   + TL/3;	
		P2p =     TL/3    + 5*TR/6   - TRR/6;
		P2n =     TR/3    + 5*TL/6   - TLL/6;
		P3p =  11*TR/6    - 7*TRR/6  + TRRR/3;
		P3n =  11*TL/6    - 7*TLL/6  + TLLL/3;
			
		IS0p = 13*pow((TLLL - 2*TLL + TL),2)/12    +  pow(( TLLL - 4*TLL + 3*TL),2)/4;
		IS0n = 13*pow((TRRR - 2*TRR + TR),2)/12    +  pow(( TRRR - 4*TRR + 3*TR),2)/4;
		IS1p = 13*pow((TLL  - 2*TL  + TR),2)/12    +  pow(( -TLL + TR),2)/4;
		IS1n = 13*pow((TRR  - 2*TR  + TL),2)/12    +  pow(( -TRR + TL),2)/4;
		IS2p = 13*pow((TL   - 2*TR  + TRR),2)/12   +  pow((-3*TL + 4*TR - TRR),2)/4;
		IS2n = 13*pow((TR   - 2*TL  + TLL),2)/12   +  pow((-3*TR + 4*TL - TLL),2)/4;
		IS3p = 13*pow((TR   - 2*TRR + TRRR),2)/12  +  pow((-5*TR + 8*TRR - 3*TRRR),2)/4;
		IS3n = 13*pow((TL   - 2*TLL + TLLL),2)/12  +  pow((-5*TL + 8*TLL - 3*TLLL),2)/4;
		
		ISPMAX = max(IS0p,   IS1p);
		ISPMAX = max(ISPMAX, IS2p);
		ISPMAX = max(ISPMAX, IS3p);
		ISNMAX = max(IS0n,   IS1n);
		ISNMAX = max(ISNMAX, IS2n);
		ISNMAX = max(ISNMAX, IS3n);
		IS3p = ISPMAX;
		IS3n = ISNMAX;
				
		ALPHA0p = gam0*(1/(eps+IS0p)); 
		ALPHA0n = gam0*(1/(eps+IS0n)); 
		ALPHA1p = gam1*(1/(eps+IS1p)); 
		ALPHA1n = gam1*(1/(eps+IS1n)); 
		ALPHA2p = gam2*(1/(eps+IS2p)); 
		ALPHA2n = gam2*(1/(eps+IS2n)); 
		ALPHA3p = gam3*(1/(eps+IS3p)); 
		ALPHA3n = gam3*(1/(eps+IS3n));  
	
		sum_ALPHAp = ALPHA0p + ALPHA1p + ALPHA2p + ALPHA3p;
		sum_ALPHAn = ALPHA0n + ALPHA1n + ALPHA2n + ALPHA3n;
	
		OMEGA0p = ALPHA0p/(sum_ALPHAp);
		OMEGA0n = ALPHA0n/(sum_ALPHAn);
		OMEGA1p = ALPHA1p/(sum_ALPHAp);
		OMEGA1n = ALPHA1n/(sum_ALPHAn);
		OMEGA2p = ALPHA2p/(sum_ALPHAp);
		OMEGA2n = ALPHA2n/(sum_ALPHAn);
		OMEGA3p = ALPHA3p/(sum_ALPHAp);
		OMEGA3n = ALPHA3n/(sum_ALPHAn);
	 
		ptype Tl = (OMEGA0p*P0p + OMEGA1p*P1p + OMEGA2p*P2p + OMEGA3p*P3p); 
		ptype Tr = (OMEGA0n*P0n + OMEGA1n*P1n + OMEGA2n*P2n + OMEGA3n*P3n); 
		
		Wl[4] = Wl[0]*R*Tl/(GAM-1.0) + 0.5*(Wl[1]*Wl[1]+Wl[2]*Wl[2]+Wl[3]*Wl[3])/Wl[0];
		Wr[4] = Wr[0]*R*Tr/(GAM-1.0) + 0.5*(Wr[1]*Wr[1]+Wr[2]*Wr[2]+Wr[3]*Wr[3])/Wr[0];	
/*
	FOR(q,5) {
		Wl = 0.5*(WL + WR); Wr = Wl;
	}
*/
}


__global__ void W2T3D(ptype *d_W0, ptype *d_W1, ptype *d_W2, ptype *d_W3, ptype *d_W4, ptype *d_T)
{
	int I = blockIdx.x*(blockDim.x) + threadIdx.x;
	d_T[I] = (GAM-1) * ( d_W4[I]/d_W0[I] - 0.5*(d_W1[I]*d_W1[I] + d_W2[I]*d_W2[I] + d_W3[I]*d_W3[I])/(d_W0[I]*d_W0[I]) ) / R;
}
/**************************************************************************************************************************
 * -----------------------------------------------------------------------------------------------------------------------*
 **************************************************************************************************************************/
